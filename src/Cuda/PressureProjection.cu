#include "hip/hip_runtime.h"
#include <nanovdb/util/GridHandle.h>
#include <nanovdb/util/SampleFromVoxels.h>

#include "../Utils/GridData.hpp"
#include "PointToGrid.cuh"
#include "Utils.cuh"


__global__ void divergence(const nanovdb::Coord* __restrict__ d_coord, float* __restrict__ d_value, const size_t npoints,
                           const nanovdb::Vec3fGrid* __restrict__ vel) {
	const size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= npoints) return;

	const auto velAccessor = vel->tree().getAccessor();
	// Linear interpolation sampler
	//const auto velSampler = nanovdb::createSampler<1>(velAccessor);
	const float dx = vel->voxelSize()[0];  // voxel spacing

	const nanovdb::Coord coord = d_coord[tid];
	//const nanovdb::Vec3f c = coord.asVec3s();

	// Compute neighbor coordinates in short vector form
	const nanovdb::Coord cxp1 = coord + nanovdb::Coord(1, 0, 0);
	const nanovdb::Coord cxm1 = coord - nanovdb::Coord(1, 0, 0);
	const nanovdb::Coord cyp1 = coord + nanovdb::Coord(0, 1, 0);
	const nanovdb::Coord cym1 = coord - nanovdb::Coord(0, 1, 0);
	const nanovdb::Coord czp1 = coord + nanovdb::Coord(0, 0, 1);
	const nanovdb::Coord czm1 = coord - nanovdb::Coord(0, 0, 1);

	const nanovdb::Vec3f vel_xm1 = velAccessor.getValue(cxm1);
	const nanovdb::Vec3f vel_xp1 = velAccessor.getValue(cxp1);
	const nanovdb::Vec3f vel_ym1 = velAccessor.getValue(cym1);
	const nanovdb::Vec3f vel_yp1 = velAccessor.getValue(cyp1);
	const nanovdb::Vec3f vel_zm1 = velAccessor.getValue(czm1);
	const nanovdb::Vec3f vel_zp1 = velAccessor.getValue(czp1);

	// Use central differencing: (f(i+1)-f(i-1)) / (2*dx)
	const float divX = (vel_xp1[0] - vel_xm1[0]) / (2.0f * dx);
	const float divY = (vel_yp1[1] - vel_ym1[1]) / (2.0f * dx);
	const float divZ = (vel_zp1[2] - vel_zm1[2]) / (2.0f * dx);

	d_value[tid] = divX + divY + divZ;
}

__global__ void pressureJacobiIteration(const nanovdb::Coord* __restrict__ d_coords, const size_t npoints,
                                        const nanovdb::FloatGrid* __restrict__ pressureGrid,
                                        const nanovdb::FloatGrid* __restrict__ divergenceGrid,
                                        const nanovdb::FloatGrid* __restrict__ newPressureGrid) {
	const size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= npoints) return;

	const auto pressureAccessor = pressureGrid->tree().getAccessor();
	const auto divergenceAccessor = divergenceGrid->tree().getAccessor();
	auto newPressureAccessor = newPressureGrid->tree().getAccessor();

	const auto pressureSampler = nanovdb::createSampler<1>(pressureAccessor);
	const auto divergenceSampler = nanovdb::createSampler<1>(divergenceAccessor);

	const nanovdb::Coord coord = d_coords[tid];
	const nanovdb::Vec3f c = coord.asVec3s();
	const float dx = pressureGrid->voxelSize()[0];

	// Neighbor coords
	const nanovdb::Coord cxp1 = coord + nanovdb::Coord(1, 0, 0);
	const nanovdb::Coord cxm1 = coord - nanovdb::Coord(1, 0, 0);
	const nanovdb::Coord cyp1 = coord + nanovdb::Coord(0, 1, 0);
	const nanovdb::Coord cym1 = coord - nanovdb::Coord(0, 1, 0);
	const nanovdb::Coord czp1 = coord + nanovdb::Coord(0, 0, 1);
	const nanovdb::Coord czm1 = coord - nanovdb::Coord(0, 0, 1);

	// Neighboring pressures
	const float p_xp1 = pressureAccessor.getValue(cxp1);
	const float p_xm1 = pressureAccessor.getValue(cxm1);
	const float p_yp1 = pressureAccessor.getValue(cyp1);
	const float p_ym1 = pressureAccessor.getValue(cym1);
	const float p_zp1 = pressureAccessor.getValue(czp1);
	const float p_zm1 = pressureAccessor.getValue(czm1);

	// Divergence at coord
	const float div = divergenceSampler(c);

	// Jacobi iteration step
	// (Sum of neighbors - div * dx^2) / 6
	const float p_new = (p_xp1 + p_xm1 + p_yp1 + p_ym1 + p_zp1 + p_zm1 - div * dx * dx) / 6.0f;

	// Write the new pressure value
	newPressureAccessor.set<nanovdb::SetVoxel<float>>(coord, p_new);
}

__global__ void subtractPressureGradient(const nanovdb::Coord* __restrict__ d_coords, const size_t npoints,
                                         const nanovdb::Vec3fGrid* __restrict__ vel, const nanovdb::FloatGrid* __restrict__ pressureGrid,
                                         const CudaResources<nanovdb::Vec3f, true> out, const float voxelSize) {
	const size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= npoints) return;

	const auto pressureSampler = nanovdb::createSampler<1>(pressureGrid->getAccessor());
	const auto velSampler = nanovdb::createSampler<1>(vel->getAccessor());

	const nanovdb::Coord coord = d_coords[tid];

	nanovdb::Vec3f grad = pressureSampler.gradient<>(coord.asVec3s());

	// Convert gradient from "per index" to "per world unit" by dividing by dx
	const float dx = voxelSize;
	grad *= (1.0f / dx);

	nanovdb::Vec3f v = velSampler(coord);
	v -= grad; // Stable fluids: u_new = u - ∇p  (if density=1 and dt=1 for simplicity)

	out.d_values[tid] = v;
}


void pressure_projection(const nanovdb::GridHandle<nanovdb::CudaDeviceBuffer>& in_vel, HNS::OpenVectorGrid& in_data,
                         HNS::OpenVectorGrid& out_data, const size_t iteration, const hipStream_t& stream) {
	using BufferT = nanovdb::CudaDeviceBuffer;

	const size_t npoints = in_data.size;
	constexpr unsigned int numThreads = 256;
	const unsigned int numBlocks = blocksPerGrid(npoints, numThreads);

	CudaResources<float, false> div_resources(npoints, stream);
	div_resources.LoadPointCoord(in_data.pCoords(), npoints, stream);

	const nanovdb::Vec3fGrid* in_vel_grid = in_vel.deviceGrid<nanovdb::Vec3f>();
	nanovdb::GridHandle<BufferT> divergence_handle = nanovdb::cudaVoxelsToGrid<float>(div_resources.d_coords, npoints, 0.2f);
	nanovdb::GridHandle<BufferT> pressure_handle_ping = nanovdb::cudaVoxelsToGrid<float>(div_resources.d_coords, npoints, 0.2f);
	nanovdb::GridHandle<BufferT> pressure_handle_pong = nanovdb::cudaVoxelsToGrid<float>(div_resources.d_coords, npoints, 0.2f);


	// Set Divergence Grid
	divergence<<<numBlocks, numThreads, 0, stream>>>(div_resources.d_coords, div_resources.d_values, npoints, in_vel_grid);

	nanovdb::FloatGrid* in_divergence = divergence_handle.deviceGrid<float>();
	set_grid_values<float, nanovdb::FloatTree, false><<<numBlocks, numThreads, 0, stream>>>(div_resources, npoints, in_divergence);

	// Compute Pressure
	nanovdb::FloatGrid* in_pressure_ping = pressure_handle_ping.deviceGrid<float>();
	nanovdb::FloatGrid* in_pressure_pong = pressure_handle_pong.deviceGrid<float>();

	zero_init_grid<<<numBlocks, numThreads, 0, stream>>>(div_resources, npoints, in_pressure_ping);
	zero_init_grid<<<numBlocks, numThreads, 0, stream>>>(div_resources, npoints, in_pressure_pong);

	for (int iter = 0; iter < iteration; ++iter) {
		pressureJacobiIteration<<<numBlocks, numThreads, 0, stream>>>(div_resources.d_coords, npoints, in_pressure_ping, in_divergence,
		                                                              in_pressure_pong);
		std::swap(in_pressure_ping, in_pressure_pong);
	}

	CudaResources<nanovdb::Vec3f, true> vel_resources(npoints, stream);

	// Subtract Pressure Gradient
	subtractPressureGradient<<<numBlocks, numThreads, 0, stream>>>(div_resources.d_coords, npoints, in_vel_grid, in_pressure_ping,
	                                                               vel_resources, 0.2);

	// copy vel back to host
	out_data.allocateCudaPinned(npoints);

	cudaCheck(hipMemcpy(out_data.pCoords(), div_resources.d_coords, npoints * sizeof(nanovdb::Coord), hipMemcpyDeviceToHost));
	cudaCheck(hipMemcpy(out_data.pValues(), vel_resources.d_values, npoints * sizeof(nanovdb::Vec3f), hipMemcpyDeviceToHost));

	// Unload data
	vel_resources.cleanup(stream);
	div_resources.cleanup(stream);
	cudaCheckError();
}

extern "C" void PressureProjection(const nanovdb::GridHandle<nanovdb::CudaDeviceBuffer>& in_vel, HNS::OpenVectorGrid& in_data,
                                   HNS::OpenVectorGrid& out_data, const size_t iteration, const hipStream_t& stream) {
	pressure_projection(in_vel, in_data, out_data, iteration, stream);
}

/*extern "C" void Divergence(const nanovdb::GridHandle<nanovdb::CudaDeviceBuffer>& in_vel, HNS::OpenVectorGrid& in_data,
                           HNS::OpenVectorGrid& out_data, const hipStream_t& stream) {
    using BufferT = nanovdb::CudaDeviceBuffer;

    const size_t npoints = in_data.size;
    constexpr unsigned int numThreads = 256;
    const unsigned int numBlocks = blocksPerGrid(npoints, numThreads);

    CudaResources<float, false> div_resources(npoints, stream);
    div_resources.LoadPointCoord(in_data.pCoords(), npoints, stream);

    const nanovdb::Vec3fGrid* in_vel_grid = in_vel.deviceGrid<nanovdb::Vec3f>();
    nanovdb::GridHandle<BufferT> divergence_handle = nanovdb::cudaVoxelsToGrid<float>(div_resources.d_coords, npoints, 0.2f);

    // Set Divergence Grid
    divergence<<<numBlocks, numThreads, 0, stream>>>(div_resources.d_coords, div_resources.d_values, npoints, in_vel_grid);

    // copy divergence back to host
    out_data.allocateCudaPinned(npoints);
    cudaCheck(hipMemcpy(out_data.pCoords(), div_resources.d_coords, npoints * sizeof(nanovdb::Coord), hipMemcpyDeviceToHost));
    cudaCheck(hipMemcpy(out_data.pValues(), div_resources.d_values, npoints * sizeof(float), hipMemcpyDeviceToHost));

    // Unload data
    div_resources.cleanup(stream);
    cudaCheckError();
}*/
#include "hip/hip_runtime.h"
#include <openvdb/Types.h>
#include <nanovdb/tools/cuda/PointsToGrid.cuh>

#include "../Utils/GridData.hpp"
#include "../Utils/Stencils.hpp"
#include "Utils.cuh"

__global__ void divergence_idx(const nanovdb::NanoGrid<nanovdb::ValueOnIndex>* domainGrid, const nanovdb::Coord* __restrict__ d_coord,
                               const nanovdb::Vec3f* __restrict__ velocityData, float* __restrict__ outDivergence, const float dx,
                               const size_t totalVoxels) {
	const size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= totalVoxels) return;

	const IndexOffsetSampler<0> idxSampler(*domainGrid);
	const auto velocitySampler = IndexSampler<nanovdb::Vec3f, 1>(idxSampler, velocityData);
	const nanovdb::Coord coord = d_coord[tid];

	if (!velocitySampler.isDataActive(coord)) {
		return;
	}

	const nanovdb::Vec3f c = coord.asVec3s();

	const float xp = velocitySampler(c + nanovdb::Vec3f(0.5f, 0.0f, 0.0f))[0];
	const float xm = velocitySampler(c - nanovdb::Vec3f(0.5f, 0.0f, 0.0f))[0];
	const float yp = velocitySampler(c + nanovdb::Vec3f(0.0f, 0.5f, 0.0f))[1];
	const float ym = velocitySampler(c - nanovdb::Vec3f(0.0f, 0.5f, 0.0f))[1];
	const float zp = velocitySampler(c + nanovdb::Vec3f(0.0f, 0.0f, 0.5f))[2];
	const float zm = velocitySampler(c - nanovdb::Vec3f(0.0f, 0.0f, 0.5f))[2];

	const float dixX = (xp - xm) / dx;
	const float dixY = (yp - ym) / dx;
	const float dixZ = (zp - zm) / dx;

	outDivergence[tid] = dixX + dixY + dixZ;
}

__global__ void redBlackGaussSeidelUpdate_idx(const nanovdb::NanoGrid<nanovdb::ValueOnIndex>* domainGrid,
                                              const nanovdb::Coord* __restrict__ d_coord, const float* __restrict__ divergence,
                                              float* __restrict__ pressure, const float dx, const size_t totalVoxels, const int color,
                                              const float omega) {
	size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= totalVoxels) return;

	const nanovdb::Coord c = d_coord[tid];
	const int i = c.x(), j = c.y(), k = c.z();

	// skip if not the correct color
	if (((i + j + k) & 1) != color) return;

	const IndexOffsetSampler<0> idxSampler(*domainGrid);
	const auto divSampler = IndexSampler<float, 1>(idxSampler, divergence);
	const auto pSampler = IndexSampler<float, 1>(idxSampler, pressure);

	// gather neighbors (assuming in range)
	const float pxp1 = pSampler(nanovdb::Coord(i + 1, j, k));
	const float pxm1 = pSampler(nanovdb::Coord(i - 1, j, k));
	const float pyp1 = pSampler(nanovdb::Coord(i, j + 1, k));
	const float pym1 = pSampler(nanovdb::Coord(i, j - 1, k));
	const float pzp1 = pSampler(nanovdb::Coord(i, j, k + 1));
	const float pzm1 = pSampler(nanovdb::Coord(i, j, k - 1));

	const float divVal = divSampler(c);

	// Standard 6-neighbor Laplacian-based update
	const float pOld = pSampler(c);
	const float pGS = (pxp1 + pxm1 + pyp1 + pym1 + pzp1 + pzm1 - divVal * dx * dx) / 6.0f;

	// SOR step
	const float pNew = pOld + omega * (pGS - pOld);

	// in-place update
	pressure[tid] = pNew;
}


__global__ void subtractPressureGradient_idx(const nanovdb::NanoGrid<nanovdb::ValueOnIndex>* domainGrid,
                                             const nanovdb::Coord* __restrict__ d_coords, const size_t totalVoxels,
                                             const nanovdb::Vec3f* __restrict__ velocity,  // velocity at faces
                                             const float* __restrict__ pressure,      // pressure at cell centers
                                             nanovdb::Vec3f* __restrict__ out,
                                             float voxelSize) {
	const size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= totalVoxels) return;

	// Accessors / Samplers
	const IndexOffsetSampler<0> idxSampler(*domainGrid);
	const auto pressureSampler = IndexSampler<float, 1>(idxSampler, pressure);
	const auto velSampler = IndexSampler<nanovdb::Vec3f, 1>(idxSampler, velocity);


	const float dx = voxelSize;

	// The cell center coordinate
	const nanovdb::Vec3f c = d_coords[tid].asVec3s();
	const nanovdb::Vec3f vel = sampleMACVelocity_idx(velSampler, c);
	nanovdb::Vec3f v;

	// For u component: Sample velocity at (i+1/2,j,k) relative to cell center
	{
		// For x-component, we're already at the face center
		const float p_left = pressureSampler(c);                                   // p(i,j,k)
		const float p_right = pressureSampler(c + nanovdb::Vec3f(1, 0.0f, 0.0f));  // p(i+1,j,k)
		v[0] = vel[0] - (p_right - p_left) / dx;
	}

	// For v component: Sample velocity at (i,j+1/2,k) relative to cell center
	{
		const float p_bottom = pressureSampler(c);                               // p(i,j,k)
		const float p_top = pressureSampler(c + nanovdb::Vec3f(0.0f, 1, 0.0f));  // p(i,j+1,k)
		v[1] = vel[1] - (p_top - p_bottom) / dx;
	}

	// For w component: Sample velocity at (i,j,k+1/2) relative to cell center
	{
		const float p_back = pressureSampler(c);                                   // p(i,j,k)
		const float p_front = pressureSampler(c + nanovdb::Vec3f(0.0f, 0.0f, 1));  // p(i,j,k+1)
		v[2] = vel[2] - (p_front - p_back) / dx;
	}

	out[tid] = v;
}

void pressure_projection_idx(HNS::GridIndexedData& data, const size_t iteration,
                             const float voxelSize, const hipStream_t& stream) {
	const size_t totalVoxels = data.size();
	constexpr int blockSize = 256;
	int numBlocks = (totalVoxels + blockSize - 1) / blockSize;

	auto* velocity = reinterpret_cast<nanovdb::Vec3f*>(data.pValues<openvdb::Vec3f>("vel"));

	if (!velocity) {
		std::cerr << "Error: velocity or divergence data is not available." << std::endl;
		return;
	}

	nanovdb::Vec3f* d_velocity = nullptr;
	nanovdb::Coord* d_coords = nullptr;
	float* d_divergence = nullptr;
	float* d_pressure = nullptr;

	hipMalloc(&d_velocity, totalVoxels * sizeof(nanovdb::Vec3f));
	hipMalloc(&d_coords, totalVoxels * sizeof(nanovdb::Coord));
	hipMalloc(&d_divergence, totalVoxels * sizeof(float));
	hipMalloc(&d_pressure, totalVoxels * sizeof(float));

	hipMemcpy(d_velocity, velocity, totalVoxels * sizeof(nanovdb::Vec3f), hipMemcpyHostToDevice);
	hipMemcpy(d_coords, data.pCoords(), totalVoxels * sizeof(nanovdb::Coord), hipMemcpyHostToDevice);

	hipMemset(d_divergence, 0, totalVoxels * sizeof(float));
	hipMemset(d_pressure, 0, totalVoxels * sizeof(float));

	hipDeviceSynchronize();

	nanovdb::GridHandle<nanovdb::cuda::DeviceBuffer> handle =
	nanovdb::tools::cuda::voxelsToGrid<nanovdb::ValueOnIndex, nanovdb::Coord*>(d_coords, data.size(), voxelSize);

	hipDeviceSynchronize();

	const auto gpuGrid = handle.deviceGrid<nanovdb::ValueOnIndex>();

	divergence_idx<<<numBlocks, blockSize, 0, stream>>>(gpuGrid, d_coords, d_velocity, d_divergence, voxelSize, totalVoxels);

	for (int iter = 0; iter < iteration; iter++) {
		// Red update
		redBlackGaussSeidelUpdate_idx<<<numBlocks, blockSize, 0, stream>>>(gpuGrid, d_coords, d_divergence, d_pressure, voxelSize,
		                                                                   totalVoxels, 0, 1.9);

		// Black update
		redBlackGaussSeidelUpdate_idx<<<numBlocks, blockSize, 0, stream>>>(gpuGrid, d_coords, d_divergence, d_pressure, voxelSize,
		                                                                   totalVoxels, 1, 1.9);
	}

	subtractPressureGradient_idx<<<numBlocks, blockSize, 0, stream>>>(gpuGrid, d_coords, totalVoxels, d_velocity, d_pressure, d_velocity, voxelSize);

	hipDeviceSynchronize();

	hipMemcpy(velocity, d_velocity, totalVoxels * sizeof(nanovdb::Vec3f), hipMemcpyDeviceToHost);

	hipFree(d_velocity);
	hipFree(d_coords);
	hipFree(d_divergence);
	hipFree(d_pressure);
}


extern "C" void Divergence_idx(HNS::GridIndexedData& data, const size_t iterations, const float voxelSize,
                               const hipStream_t& stream) {
	pressure_projection_idx(data, iterations, voxelSize, stream);
}
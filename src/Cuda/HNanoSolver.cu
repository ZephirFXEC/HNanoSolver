#include <openvdb/Types.h>

#include "../Utils/GridData.hpp"
#include "../Utils/Stencils.hpp"
#include "Kernels.cuh"
#include "nanovdb/NanoVDB.h"


void Compute(HNS::GridIndexedData& data, const nanovdb::GridHandle<nanovdb::cuda::DeviceBuffer>& handle, const int iteration,
             const float dt, const float voxelSize, const hipStream_t& stream) {
	const size_t totalVoxels = data.size();
	const float inv_voxelSize = 1.0f / voxelSize;

	// Get velocity block (assuming exactly one Vec3f block)
	const auto vec3fBlocks = data.getBlocksOfType<openvdb::Vec3f>();
	if (vec3fBlocks.size() != 1) {
		throw std::runtime_error("Expected exactly one Vec3f block (velocity)");
	}

	nanovdb::Vec3f* velocity = reinterpret_cast<nanovdb::Vec3f*>(data.pValues<openvdb::Vec3f>(vec3fBlocks[0]));
	if (!velocity) {
		throw std::runtime_error("Velocity data not found");
	}

	// Get all float blocks (density, temperature, fuel, etc.)
	const auto floatBlocks = data.getBlocksOfType<float>();
	if (floatBlocks.empty()) {
		throw std::runtime_error("No float blocks found");
	}

	// Create CUDA resources
	std::vector<hipStream_t> streams(floatBlocks.size());
	std::vector<float*> hostPointers(floatBlocks.size());
	std::vector<float*> d_inputs(floatBlocks.size());
	std::vector<float*> d_outputs(floatBlocks.size());

	// Initialize resources for each float block
	for (size_t i = 0; i < floatBlocks.size(); i++) {
		hipStreamCreate(&streams[i]);

		auto* host_ptr = data.pValues<float>(floatBlocks[i]);
		if (!host_ptr) {
			throw std::runtime_error("Block '" + floatBlocks[i] + "' not found or type mismatch");
		}
		hostPointers[i] = host_ptr;

		hipMalloc(&d_inputs[i], totalVoxels * sizeof(float));
		hipMalloc(&d_outputs[i], totalVoxels * sizeof(float));

		// Copy all scalar fields to device (fixing the bug where only the first field was copied)
		hipMemcpyAsync(d_inputs[i], hostPointers[i], totalVoxels * sizeof(float), hipMemcpyHostToDevice, streams[i]);
	}

	// Allocate and initialize device memory
	nanovdb::Vec3f* d_velocity = nullptr;
	nanovdb::Coord* d_coords = nullptr;
	nanovdb::Vec3f* d_outVel = nullptr;
	float* d_divergence = nullptr;
	float* d_pressure = nullptr;

	hipMalloc(&d_velocity, totalVoxels * sizeof(nanovdb::Vec3f));
	hipMalloc(&d_coords, totalVoxels * sizeof(nanovdb::Coord));
	hipMalloc(&d_outVel, totalVoxels * sizeof(nanovdb::Vec3f));
	hipMalloc(&d_divergence, totalVoxels * sizeof(float));
	hipMalloc(&d_pressure, totalVoxels * sizeof(float));

	hipMemsetAsync(d_outVel, 0, totalVoxels * sizeof(nanovdb::Vec3f), stream);
	hipMemsetAsync(d_divergence, 0, totalVoxels * sizeof(float), stream);
	hipMemsetAsync(d_pressure, 0, totalVoxels * sizeof(float), stream);

	hipMemcpyAsync(d_velocity, velocity, totalVoxels * sizeof(nanovdb::Vec3f), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(d_coords, data.pCoords(), totalVoxels * sizeof(nanovdb::Coord), hipMemcpyHostToDevice, stream);

	// Create grid and prepare for kernel launch
	const auto gpuGrid = handle.deviceGrid<nanovdb::ValueOnIndex>();

	// Calculate optimal kernel launch parameters
	int deviceId, blockSize, minGridSize;
	hipGetDevice(&deviceId);
	hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, advect_vector, 0, 0);
	const int gridSize = (totalVoxels + blockSize - 1) / blockSize;


	// Simulation pipeline with synchronization points

	// Step 1: Advect velocity field
	advect_vector<<<gridSize, blockSize, 0, stream>>>(gpuGrid, d_coords, d_velocity, d_outVel, totalVoxels, dt, voxelSize);
	hipMemcpyAsync(d_velocity, d_outVel, totalVoxels * sizeof(nanovdb::Vec3f), hipMemcpyDeviceToDevice, stream);

	// Step 2: Apply buoyancy forces
	vel_y_density<<<gridSize, blockSize, 0, stream>>>(gpuGrid, d_coords, d_velocity, d_inputs[0], d_outVel, totalVoxels);
	hipMemcpyAsync(d_velocity, d_outVel, totalVoxels * sizeof(nanovdb::Vec3f), hipMemcpyDeviceToDevice, stream);

	// Step 3: Calculate velocity field divergence
	divergence<<<gridSize, blockSize, 0, stream>>>(gpuGrid, d_coords, d_velocity, d_divergence, voxelSize, totalVoxels);

	// Step 4: Pressure solver (Red-black Gauss-Seidel iterations)
	constexpr float omega = 1.9f;  // SOR relaxation parameter
	for (int iter = 0; iter < iteration; ++iter) {
		redBlackGaussSeidelUpdate<<<gridSize, blockSize, 0, stream>>>(gpuGrid, d_coords, d_divergence, d_pressure, voxelSize, totalVoxels,
		                                                              0, omega);

		redBlackGaussSeidelUpdate<<<gridSize, blockSize, 0, stream>>>(gpuGrid, d_coords, d_divergence, d_pressure, voxelSize, totalVoxels,
		                                                              1, omega);
	}

	// Step 5: Apply pressure gradient to enforce incompressibility
	subtractPressureGradient<<<gridSize, blockSize, 0, stream>>>(gpuGrid, d_coords, totalVoxels, d_velocity, d_pressure, d_velocity,
	                                                             voxelSize);

	// Sync before advecting scalar fields
	hipStreamSynchronize(stream);

	// Step 6: Advect all scalar fields in parallel using individual streams
	for (size_t i = 0; i < floatBlocks.size(); ++i) {
		advect_scalar<<<gridSize, blockSize, 0, streams[i]>>>(gpuGrid, d_coords, d_velocity, d_inputs[i], d_outputs[i], totalVoxels, dt,
		                                                      inv_voxelSize);

		// Copy results back to host
		hipMemcpyAsync(hostPointers[i], d_outputs[i], totalVoxels * sizeof(float), hipMemcpyDeviceToHost, streams[i]);
	}

	// Synchronize all streams
	for (auto& s : streams) {
		hipStreamSynchronize(s);
	}

	hipMemcpy(velocity, d_velocity, totalVoxels * sizeof(nanovdb::Vec3f), hipMemcpyDeviceToHost);

	// Clean up all allocated resources
	// Free device memory
	hipFree(d_velocity);
	hipFree(d_coords);
	hipFree(d_outVel);
	hipFree(d_divergence);
	hipFree(d_pressure);

	// Free scalar field resources
	for (size_t i = 0; i < floatBlocks.size(); ++i) {
		hipFree(d_inputs[i]);
		hipFree(d_outputs[i]);
		hipStreamDestroy(streams[i]);
	}
}

void create_index_grid(HNS::GridIndexedData& data, nanovdb::GridHandle<nanovdb::cuda::DeviceBuffer>& handle, const float voxelSize) {
	const auto* h_coords = data.pCoords();
	nanovdb::Coord* d_coords = nullptr;
	hipMalloc(&d_coords, data.size() * sizeof(nanovdb::Coord));
	hipMemcpy(d_coords, h_coords, data.size() * sizeof(nanovdb::Coord), hipMemcpyHostToDevice);

	handle = nanovdb::tools::cuda::voxelsToGrid<nanovdb::ValueOnIndex, nanovdb::Coord*>(d_coords, data.size(), voxelSize);

	hipFree(d_coords);
}

extern "C" void CreateIndexGrid(HNS::GridIndexedData& data, nanovdb::GridHandle<nanovdb::cuda::DeviceBuffer>& handle,
                                const float voxelSize) {
	create_index_grid(data, handle, voxelSize);
}


extern "C" void Compute_Sim(HNS::GridIndexedData& data, const nanovdb::GridHandle<nanovdb::cuda::DeviceBuffer>& handle, const int iteration,
                            const float dt, const float voxelSize, const hipStream_t& stream) {
	Compute(data, handle, iteration, dt, voxelSize, stream);
}
#include "hip/hip_runtime.h"
#include <nanovdb/NanoVDB.h>
#include <nanovdb/util/SampleFromVoxels.h>

#include <cuda/std/cmath>

#include "../Utils/GridData.hpp"
#include "HNanoGrid/HNanoGrid.cuh"
#include "PointToGrid.cuh"


template <typename T, typename U = std::conditional_t<std::is_same_v<T, float>, nanovdb::FloatTree, nanovdb::Vec3fTree>>
__global__ void advect(const CudaResources<T, true> resources, const size_t npoints, const float dt, const float voxelSize,
                       const nanovdb::Vec3fGrid* __restrict__ vel_grid, const nanovdb::Grid<U>* __restrict__ d_grid) {
	// Precompute constants
	const float inv_voxelSize = 1.0f / voxelSize;
	const float scaled_dt = dt * inv_voxelSize;

	const size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= npoints) return;

	auto accessor = d_grid->tree().getAccessor();
	const auto velAccessor = vel_grid->tree().getAccessor();
	const auto velSampler = nanovdb::createSampler<1>(velAccessor);
	auto valueSampler = nanovdb::createSampler<1>(accessor);

	const nanovdb::Coord ijk = resources.d_coords[tid];
	const T value = resources.d_values[tid];

	if (!accessor.isActive(ijk)) {
		return;
	}

	const nanovdb::Vec3f voxelCoordf = ijk.asVec3s();

	// Forward step
	const nanovdb::Vec3f velocity = velSampler(voxelCoordf);
	const nanovdb::Vec3f forward_pos = voxelCoordf - velocity * scaled_dt;
	const T value_forward = valueSampler(forward_pos);

	// Backward step
	const nanovdb::Vec3f back_velocity = velSampler(forward_pos);
	const nanovdb::Vec3f back_pos = voxelCoordf + back_velocity * scaled_dt;
	const T value_backward = valueSampler(back_pos);

	// Error estimation and correction
	const T error = computeError(value, value_backward);
	T value_corrected = value_forward + error;

	const T max_correction = computeMaxCorrection(value_forward, value);
	value_corrected = clampValue(value_corrected, value_forward - max_correction, value_forward + max_correction);

	constexpr float blend_factor = 0.8f;
	T new_value = lerp(value_forward, value_corrected, blend_factor);

	new_value = enforceNonNegative(new_value);

	// Store the new value
	resources.d_temp_values[tid] = new_value;
}


void advect_points_to_grid_f(HNS::OpenFloatGrid& in_data, const nanovdb::Vec3fGrid* vel_grid, HNS::NanoFloatGrid& out_data,
                             const float voxelSize, const float dt, const hipStream_t& stream) {
	const size_t npoints = in_data.size;

	cudaCheck(hipHostRegister(in_data.pCoords(), npoints * sizeof(openvdb::Coord), hipHostRegisterDefault));
	cudaCheck(hipHostRegister(in_data.pValues(), npoints * sizeof(float), hipHostRegisterDefault));

	CudaResources<float, true> resources(npoints, stream);

	nanovdb::GridHandle<nanovdb::CudaDeviceBuffer> handle;
	pointToTopologyToDevice<float, true>(resources, in_data.pCoords(), npoints, voxelSize, handle, stream);
	fillTopology<float, float, true>(resources, in_data.pValues(), npoints, handle, stream);

	constexpr unsigned int numThreads = 256;
	const unsigned int numBlocks = blocksPerGrid(npoints, numThreads);

	const nanovdb::FloatGrid* d_grid = handle.deviceGrid<float>();
	advect<float><<<numBlocks, numThreads, 0, stream>>>(resources, npoints, dt, voxelSize, vel_grid, d_grid);

	out_data.allocateStandard(npoints);

	cudaCheck(hipHostRegister(out_data.pCoords(), npoints * sizeof(nanovdb::Coord), hipHostRegisterDefault));
	cudaCheck(hipHostRegister(out_data.pValues(), npoints * sizeof(float), hipHostRegisterDefault));

	resources.UnloadPointData(out_data, stream);

	cudaCheck(hipHostUnregister(in_data.pCoords()));
	cudaCheck(hipHostUnregister(in_data.pValues()));
	cudaCheck(hipHostUnregister(out_data.pCoords()));
	cudaCheck(hipHostUnregister(out_data.pValues()));

	resources.cleanup(stream);
}

void advect_points_to_grid_v(HNS::OpenVectorGrid& in_data, HNS::NanoVectorGrid& out_data, const float voxelSize, const float dt,
                             const hipStream_t& stream) {
	const size_t npoints = in_data.size;

	cudaCheck(hipHostRegister(in_data.pCoords(), npoints * sizeof(openvdb::Coord), hipHostRegisterDefault));
	cudaCheck(hipHostRegister(in_data.pValues(), npoints * sizeof(openvdb::Vec3f), hipHostRegisterDefault));

	CudaResources<nanovdb::Vec3f, true> resources(npoints, stream);

	nanovdb::GridHandle<nanovdb::CudaDeviceBuffer> handle;
	pointToTopologyToDevice<nanovdb::Vec3f, true>(resources, in_data.pCoords(), npoints, voxelSize, handle, stream);
	fillTopology<nanovdb::Vec3f, openvdb::Vec3f, true>(resources, in_data.pValues(), npoints, handle, stream);

	constexpr unsigned int numThreads = 256;
	const unsigned int numBlocks = blocksPerGrid(npoints, numThreads);

	const nanovdb::Vec3fGrid* d_grid = handle.deviceGrid<nanovdb::Vec3f>();
	advect<nanovdb::Vec3f><<<numBlocks, numThreads, 0, stream>>>(resources, npoints, dt, voxelSize, d_grid, d_grid);

	out_data.allocateStandard(npoints);

	cudaCheck(hipHostRegister(out_data.pCoords(), npoints * sizeof(nanovdb::Coord), hipHostRegisterDefault));
	cudaCheck(hipHostRegister(out_data.pValues(), npoints * sizeof(nanovdb::Vec3f), hipHostRegisterDefault));

	resources.UnloadPointData(out_data, stream);

	cudaCheck(hipHostUnregister(in_data.pCoords()));
	cudaCheck(hipHostUnregister(in_data.pValues()));
	cudaCheck(hipHostUnregister(out_data.pCoords()));
	cudaCheck(hipHostUnregister(out_data.pValues()));

	resources.cleanup(stream);
}


extern "C" void AdvectFloat(HNS::OpenFloatGrid& in_data, const nanovdb::Vec3fGrid* vel_grid, HNS::NanoFloatGrid& out_data,
                            const float voxelSize, const float dt, const hipStream_t& stream) {
	advect_points_to_grid_f(in_data, vel_grid, out_data, voxelSize, dt, stream);
}

extern "C" void AdvectVector(HNS::OpenVectorGrid& in_data, HNS::NanoVectorGrid& out_data, const float voxelSize, const float dt,
                             const hipStream_t& stream) {
	advect_points_to_grid_v(in_data, out_data, voxelSize, dt, stream);
}
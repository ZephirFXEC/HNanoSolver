#include "hip/hip_runtime.h"
#include <nanovdb/NanoVDB.h>

#include <cuda/std/cmath>
#include "../Utils/GridData.hpp"
#include "../Utils/Stencils.hpp"
#include <openvdb/Types.h>
#include <nanovdb/tools/cuda/PointsToGrid.cuh>

__global__ void advect_idx(
	const nanovdb::NanoGrid<nanovdb::ValueOnIndex>* domainGrid,
	const nanovdb::Coord* __restrict__ coords,
	const nanovdb::Vec3f* __restrict__ velocityData,
	const float* __restrict__ inData,
	float* __restrict__ outData,
	const size_t totalVoxels,
	const float dt,
	const float voxelSize)
{
	const uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= totalVoxels) return;

	const IndexOffsetSampler<0> idxSampler(*domainGrid);
	const auto densitySampler = IndexSampler<float, 1>(idxSampler, inData);

	const auto velocitySampler = IndexSampler<nanovdb::Vec3f, 1>(idxSampler, velocityData);

	const nanovdb::Coord coord = coords[idx];
	const nanovdb::Vec3f velocity = velocitySampler(coord);

	const nanovdb::Vec3f displacedPos = coord.asVec3s() - velocity * dt / voxelSize;

	outData[idx] = densitySampler(displacedPos);
}


__global__ void advect_idx(
	const nanovdb::NanoGrid<nanovdb::ValueOnIndex>* domainGrid,
	const nanovdb::Coord* __restrict__ coords,
	const nanovdb::Vec3f* __restrict__ velocityData,
	nanovdb::Vec3f* __restrict__ outVelocity,
	const size_t totalVoxels,
	const float dt,
	const float voxelSize)
{
	const uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= totalVoxels) return;

	const IndexOffsetSampler<0> idxSampler(*domainGrid);

	const auto velocitySampler = IndexSampler<nanovdb::Vec3f, 1>(idxSampler, velocityData);

	const nanovdb::Coord coord = coords[idx];
	const nanovdb::Vec3f velocity = velocitySampler(coord);

	const nanovdb::Vec3f displacedPos = coord.asVec3s() - velocity * dt / voxelSize;

	outVelocity[idx] = velocitySampler(displacedPos);
}


void advect_index_grid(HNS::GridIndexedData& data, const float dt,
                       const float voxelSize, const hipStream_t& stream) {
	const size_t totalVoxels = data.size();

	const nanovdb::Vec3f* velocity = reinterpret_cast<nanovdb::Vec3f*>(data.pValues<openvdb::Vec3f>("vel"));
	auto* density = data.pValues<float>("density");
	auto* temperature = data.pValues<float>("temperature");
	auto* fuel = data.pValues<float>("fuel");

	if (!velocity || !density || !temperature || !fuel) {
		throw std::runtime_error("Density data not found in the grid.");
	}

	// Allocate device memory.
	nanovdb::Vec3f* d_velocity = nullptr;
	nanovdb::Coord* d_coords = nullptr;

	float* d_density = nullptr;
	float* d_temperature = nullptr;
	float* d_fuel = nullptr;

	hipMalloc(&d_velocity, totalVoxels * sizeof(nanovdb::Vec3f));
	hipMalloc(&d_coords, totalVoxels * sizeof(nanovdb::Coord));
	hipMalloc(&d_density, totalVoxels * sizeof(float));
	hipMalloc(&d_temperature, totalVoxels * sizeof(float));
	hipMalloc(&d_fuel, totalVoxels * sizeof(float));

	hipDeviceSynchronize();

	hipMemcpy(d_coords, data.pCoords(), totalVoxels * sizeof(nanovdb::Coord), hipMemcpyHostToDevice);
	hipMemcpy(d_density, density, totalVoxels * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_temperature, temperature, totalVoxels * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_fuel, fuel, totalVoxels * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_velocity, velocity, totalVoxels * sizeof(nanovdb::Vec3f), hipMemcpyHostToDevice);

	// Allocate device memory for the output density.
	float* d_outDensity = nullptr;
	float* d_outTemperature = nullptr;
	float* d_outFuel = nullptr;

	hipMalloc(&d_outDensity, totalVoxels * sizeof(float));
	hipMalloc(&d_outTemperature, totalVoxels * sizeof(float));
	hipMalloc(&d_outFuel, totalVoxels * sizeof(float));

	hipDeviceSynchronize();

	nanovdb::GridHandle<nanovdb::cuda::DeviceBuffer> handle =
	nanovdb::tools::cuda::voxelsToGrid<nanovdb::ValueOnIndex, nanovdb::Coord*>(d_coords, data.size(), voxelSize);

	hipDeviceSynchronize();

	const auto gpuGrid = handle.deviceGrid<nanovdb::ValueOnIndex>();

	hipDeviceSynchronize();

	constexpr int blockSize = 256;
	int numBlocks = (totalVoxels + blockSize - 1) / blockSize;

	advect_idx<<<numBlocks, blockSize, 0, stream>>>(gpuGrid, d_coords, d_velocity, d_density, d_outDensity, totalVoxels, dt, voxelSize);
	advect_idx<<<numBlocks, blockSize, 0, stream>>>(gpuGrid, d_coords, d_velocity, d_temperature, d_outTemperature, totalVoxels, dt, voxelSize);
	advect_idx<<<numBlocks, blockSize, 0, stream>>>(gpuGrid, d_coords, d_velocity, d_fuel, d_outFuel, totalVoxels, dt, voxelSize);

	hipDeviceSynchronize();

	hipMemcpy(density, d_outDensity, totalVoxels * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(temperature, d_outTemperature, totalVoxels * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(fuel, d_outFuel, totalVoxels * sizeof(float), hipMemcpyDeviceToHost);

	hipDeviceSynchronize();

	// Free the allocated device memory.
	hipFree(d_velocity);
	hipFree(d_coords);
	hipFree(d_density);
	hipFree(d_outDensity);
	hipFree(d_temperature);
	hipFree(d_outTemperature);
	hipFree(d_fuel);
	hipFree(d_outFuel);
}


void advect_index_grid_v(HNS::GridIndexedData& data, const float dt,
                       const float voxelSize, const hipStream_t& stream) {
	const size_t totalVoxels = data.size();

	nanovdb::Vec3f* velocity = reinterpret_cast<nanovdb::Vec3f*>(data.pValues<openvdb::Vec3f>("vel"));

	nanovdb::Vec3f* d_velocity = nullptr;
	hipMalloc(&d_velocity, totalVoxels * sizeof(nanovdb::Vec3f));
	// Use synchronous copy for non-pinned memory
	hipMemcpy(d_velocity, velocity, totalVoxels * sizeof(nanovdb::Vec3f), hipMemcpyHostToDevice);

	// Allocate device memory for voxel coordinates.
	nanovdb::Coord* d_coords = nullptr;
	hipMalloc(&d_coords, totalVoxels * sizeof(nanovdb::Coord));
	hipMemcpy(d_coords, data.pCoords(), totalVoxels * sizeof(nanovdb::Coord), hipMemcpyHostToDevice);

	// Allocate device memory for the output density.
	nanovdb::Vec3f* d_outVel = nullptr;
	hipMalloc(&d_outVel, totalVoxels * sizeof(nanovdb::Vec3f));
	hipDeviceSynchronize();

	nanovdb::GridHandle<nanovdb::cuda::DeviceBuffer> handle =
	nanovdb::tools::cuda::voxelsToGrid<nanovdb::ValueOnIndex, nanovdb::Coord*>(d_coords, data.size(), voxelSize);
	const auto gpuGrid = handle.deviceGrid<nanovdb::ValueOnIndex>();
	hipDeviceSynchronize();

	constexpr int blockSize = 256;
	int numBlocks = (totalVoxels + blockSize - 1) / blockSize;
	advect_idx<<<numBlocks, blockSize, 0, stream>>>(gpuGrid, d_coords, d_velocity, d_outVel, totalVoxels, dt, voxelSize);

	// Make sure kernel is finished before copying back
	hipDeviceSynchronize();

	// Use synchronous copy for non-pinned memory
	hipMemcpy(velocity, d_outVel, totalVoxels * sizeof(nanovdb::Vec3f), hipMemcpyDeviceToHost);

	// Free the allocated device memory.
	hipFree(d_velocity);
	hipFree(d_outVel);
	hipFree(d_coords);
}


extern "C" void AdvectIndexGrid(HNS::GridIndexedData& data,
                                const float dt, const float voxelSize, const hipStream_t& stream) {
	advect_index_grid(data, dt, voxelSize, stream);
}

extern "C" void AdvectIndexGridVelocity(HNS::GridIndexedData& data, const float dt,
				const float voxelSize, const hipStream_t& stream) {
	advect_index_grid_v(data, dt, voxelSize, stream);
}
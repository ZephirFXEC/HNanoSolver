#include "hip/hip_runtime.h"
#include <cuda/std/__algorithm/clamp.h>
#include <nanovdb/NanoVDB.h>
#include <nanovdb/util/SampleFromVoxels.h>

#include <cuda/std/cmath>

#include "../Utils/GridData.hpp"
#include "HNanoGrid/HNanoGrid.cuh"
#include "Utils.cuh"


extern "C" void advect_points_to_grid_f(HNS::OpenFloatGrid& in_data, const nanovdb::Vec3fGrid* vel_grid,
                                        HNS::NanoFloatGrid& out_data, const float voxelSize, const float dt,
                                        const hipStream_t& stream) {
	const size_t npoints = in_data.size;

	cudaCheck(hipHostRegister(in_data.pCoords(), npoints * sizeof(openvdb::Coord), hipHostRegisterDefault));
	cudaCheck(hipHostRegister(in_data.pValues(), npoints * sizeof(float), hipHostRegisterDefault));

	CudaResources<float> resources(npoints, stream);
	resources.LoadPointData<float>(in_data, stream);

	cudaCheck(hipStreamWaitEvent(stream, resources.CoordBeenCopied, 0));

	auto handle = nanovdb::cudaVoxelsToGrid<float>(resources.d_coords, npoints, voxelSize);
	nanovdb::FloatGrid* d_grid = handle.deviceGrid<float>();

	constexpr unsigned int numThreads = 256;
	const unsigned int numBlocks = blocksPerGrid(npoints, numThreads);

	cudaCheck(hipStreamWaitEvent(stream, resources.ValueBeenCopied, 0));
	lambdaKernel<<<numBlocks, numThreads, 0, stream>>>(npoints, [=] __device__(const size_t tid) {
		const auto accessor = d_grid->tree().getAccessor();
		accessor.set<nanovdb::SetVoxel<float>>(resources.d_coords[tid], resources.d_values[tid]);
	});

	lambdaKernel<<<numBlocks, numThreads, 0, stream>>>(npoints, [=] __device__(const size_t tid) {
		const nanovdb::Coord& ijk = resources.d_coords[tid];
		const float& density = resources.d_values[tid];

		const auto accessor = d_grid->tree().getAccessor();

		if (accessor.isActive(ijk)) {
			const auto velAccessor = vel_grid->tree().getAccessor();
			const auto velSampler = nanovdb::createSampler<1>(velAccessor);
			const auto denSampler = nanovdb::createSampler<1>(accessor);

			const nanovdb::Vec3f voxelCoordf = ijk.asVec3s();
			const float inv_voxelSize = 1.0f / voxelSize;

			// Forward step
			const nanovdb::Vec3f velocity = velSampler(voxelCoordf);
			const nanovdb::Vec3f forward_pos = voxelCoordf - velocity * (dt * inv_voxelSize);
			const float d_forward = denSampler(forward_pos);

			// Backward step
			const nanovdb::Vec3f back_pos = voxelCoordf + velSampler(forward_pos) * (dt * inv_voxelSize);
			const float d_backward = denSampler(back_pos);

			// Error estimation and correction
			const float error = 0.5f * (density - d_backward);
			float d_corrected = d_forward + error;

			// Limit the correction based on the neighborhood of the forward position
			const float max_correction = 0.5f * fabsf(d_forward - density);
			d_corrected = __saturatef((d_corrected - d_forward + max_correction) * (1.0f / (2.0f * max_correction))) *
			                  (2.0f * max_correction) +
			              d_forward - max_correction;

			// Final advection (blend between semi-Lagrangian and BFECC result)
			constexpr float blend_factor = 0.8f;
			float new_density = __fmaf_rn(blend_factor, d_corrected - d_forward, d_forward);

			// Ensure non-negativity
			new_density = fmaxf(0.0f, new_density);

			resources.d_temp_values[tid] = new_density;
		}
	});

	out_data.allocateStandard(npoints);

	cudaCheck(hipHostRegister(out_data.pCoords(), npoints * sizeof(nanovdb::Coord), hipHostRegisterDefault));
	cudaCheck(hipHostRegister(out_data.pValues(), npoints * sizeof(float), hipHostRegisterDefault));

	resources.UnloadPointData(out_data, stream);

	cudaCheck(hipHostUnregister(in_data.pCoords()));
	cudaCheck(hipHostUnregister(in_data.pValues()));
	cudaCheck(hipHostUnregister(out_data.pCoords()));
	cudaCheck(hipHostUnregister(out_data.pValues()));

	resources.cleanup(stream);
}

extern "C" void advect_points_to_grid_v(HNS::OpenVectorGrid& in_data, HNS::NanoVectorGrid& out_data, const float voxelSize, const float dt,
                                        const hipStream_t& stream) {
	const size_t npoints = in_data.size;

	cudaCheck(hipHostRegister(in_data.pCoords(), npoints * sizeof(openvdb::Coord), hipHostRegisterDefault));
	cudaCheck(hipHostRegister(in_data.pValues(), npoints * sizeof(openvdb::Vec3f), hipHostRegisterDefault));

	CudaResources<nanovdb::Vec3f> resources(npoints, stream);
	resources.LoadPointData<openvdb::Vec3f>(in_data, stream);

	cudaCheck(hipStreamWaitEvent(stream, resources.CoordBeenCopied, 0));
	auto handle = nanovdb::cudaVoxelsToGrid<nanovdb::Vec3f>(resources.d_coords, npoints, voxelSize);
	nanovdb::Vec3fGrid* d_grid = handle.deviceGrid<nanovdb::Vec3f>();

	constexpr unsigned int numThreads = 256;
	const unsigned int numBlocks = blocksPerGrid(npoints, numThreads);

	cudaCheck(hipStreamWaitEvent(stream, resources.ValueBeenCopied, 0));

	lambdaKernel<<<numBlocks, numThreads, 0, stream>>>(npoints, [=] __device__(const size_t tid) {
		const auto accessor = d_grid->tree().getAccessor();
		accessor.set<nanovdb::SetVoxel<nanovdb::Vec3f>>(resources.d_coords[tid], resources.d_values[tid]);
	}); cudaCheckError();

	lambdaKernel<<<numBlocks, numThreads, 0, stream>>>(npoints, [=] __device__(const size_t tid) {
		const nanovdb::Coord& ijk = resources.d_coords[tid];
		const nanovdb::Vec3f& velocity = resources.d_values[tid];
		const auto velAccessor = d_grid->tree().getAccessor();

		if (!velAccessor.isActive(ijk)) {
			return;
		}

		const auto velSampler = nanovdb::createSampler<1>(velAccessor);

		const float inv_voxelSize = 1.0f / voxelSize;
		const nanovdb::Vec3f voxelCoordf = ijk.asVec3s();
		const nanovdb::Vec3f scaled_dt_velocity = velocity * (dt * inv_voxelSize);

		// Perform forward and backward advection using velocity
		const nanovdb::Vec3f forward_pos = voxelCoordf - scaled_dt_velocity;
		const nanovdb::Vec3f backward_pos = voxelCoordf + scaled_dt_velocity;

		const nanovdb::Vec3f v_forward = velSampler(forward_pos);
		const nanovdb::Vec3f v_backward = velSampler(backward_pos);

		// Error estimation and correction
		const nanovdb::Vec3f error = 0.5f * (velocity - v_backward);
		nanovdb::Vec3f v_corrected = v_forward + error;

		nanovdb::Vec3f max_correction;
		max_correction[0] = cuda::std::abs(0.5f * (v_forward[0] - velocity[0]));
		max_correction[1] = cuda::std::abs(0.5f * (v_forward[1] - velocity[1]));
		max_correction[2] = cuda::std::abs(0.5f * (v_forward[2] - velocity[2]));

		v_corrected[0] = cuda::std::clamp(v_corrected[0], v_forward[0] - max_correction[0], v_forward[0] + max_correction[0]);
		v_corrected[1] = cuda::std::clamp(v_corrected[1], v_forward[1] - max_correction[1], v_forward[1] + max_correction[1]);
		v_corrected[2] = cuda::std::clamp(v_corrected[2], v_forward[2] - max_correction[2], v_forward[2] + max_correction[2]);


		constexpr float blend_factor = 0.8f;  // Adjust this value between 0 and 1
		nanovdb::Vec3f new_velocity;
		new_velocity[0] = lerp(v_forward[0], v_corrected[0], blend_factor);
		new_velocity[1] = lerp(v_forward[1], v_corrected[1], blend_factor);
		new_velocity[2] = lerp(v_forward[2], v_corrected[2], blend_factor);

		// Store the new velocity
		resources.d_temp_values[tid] = new_velocity;
	});
	cudaCheckError();

	out_data.allocateStandard(npoints);

	cudaCheck(hipHostRegister(out_data.pCoords(), npoints * sizeof(nanovdb::Coord), hipHostRegisterDefault));
	cudaCheck(hipHostRegister(out_data.pValues(), npoints * sizeof(nanovdb::Vec3f), hipHostRegisterDefault));

	resources.UnloadPointData(out_data, stream);

	cudaCheck(hipHostUnregister(in_data.pCoords()));
	cudaCheck(hipHostUnregister(in_data.pValues()));
	cudaCheck(hipHostUnregister(out_data.pCoords()));
	cudaCheck(hipHostUnregister(out_data.pValues()));

	resources.cleanup(stream);
}
#include "hip/hip_runtime.h"
#include <nanovdb/NanoVDB.h>

#include <cuda/std/cmath>
#include "../Utils/GridData.hpp"
#include "../Utils/Stencils.hpp"
#include <openvdb/Types.h>
#include <nanovdb/tools/cuda/PointsToGrid.cuh>
#include "Utils.cuh"

__global__ void advect_idx(
	const nanovdb::NanoGrid<nanovdb::ValueOnIndex>* domainGrid,
	const nanovdb::Coord* __restrict__ coords,
	const nanovdb::Vec3f* __restrict__ velocityData,
	const float* __restrict__ inData,
	float* __restrict__ outData,
	const size_t totalVoxels,
	const float dt,
	const float voxelSize)
{
	const uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= totalVoxels) return;

	const float inv_voxelSize = 1.0f / voxelSize;
	const float scaled_dt = dt * inv_voxelSize;

	const IndexOffsetSampler<0> idxSampler(*domainGrid);
	const auto velocitySampler = IndexSampler<nanovdb::Vec3f, 1>(idxSampler, velocityData);
	const auto dataSampler = IndexSampler<float, 1>(idxSampler, inData);

	const nanovdb::Coord coord = coords[idx];
	const nanovdb::Vec3f pos = coord.asVec3s();

	const float original = dataSampler(coord);
	// -------------------------------------------
	// Forward step (semi-Lagrangian):
	// Trace backward in time to find donor cell.
	// velocity at voxelCoordf (MAC-sampled)
	const nanovdb::Vec3f velocity = velocitySampler(pos);
	const nanovdb::Vec3f forward_pos = pos - velocity * scaled_dt;
	const float value_forward = dataSampler(forward_pos);


	// -------------------------------------------
	// Backward step for BFECC:
	// From the forward_pos, integrate forward dt again:
	const nanovdb::Vec3f back_velocity = velocitySampler(forward_pos);
	const nanovdb::Vec3f back_pos = pos + back_velocity * scaled_dt;
	const float value_backward = dataSampler(back_pos);


	// Error estimation and correction
	const float error = computeError(original, value_backward);
	float value_corrected = value_forward + error;
	const float max_correction = computeMaxCorrection(value_forward, original);
	value_corrected = clampValue(value_corrected, value_forward - max_correction, value_forward + max_correction);
	value_corrected = enforceNonNegative(value_corrected);


	// Store the new value
	outData[idx] = value_corrected;
}


__global__ void advect_idx(
	const nanovdb::NanoGrid<nanovdb::ValueOnIndex>* domainGrid,
	const nanovdb::Coord* __restrict__ coords,
	const nanovdb::Vec3f* __restrict__ velocityData,
	nanovdb::Vec3f* __restrict__ outVelocity,
	const size_t totalVoxels,
	const float dt,
	const float voxelSize)
{
	const uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= totalVoxels) return;

	const float inv_voxelSize = 1.0f / voxelSize;
	const float scaled_dt = dt * inv_voxelSize;

	const IndexOffsetSampler<0> idxSampler(*domainGrid);
	const auto velocitySampler = IndexSampler<nanovdb::Vec3f, 1>(idxSampler, velocityData);

	const nanovdb::Coord coord = coords[idx];
	const nanovdb::Vec3f pos = coord.asVec3s();

	const nanovdb::Vec3f velocity = velocitySampler(coord);
	const nanovdb::Vec3f forward_pos = pos - velocity * scaled_dt;
	const nanovdb::Vec3f value_forward = velocitySampler(forward_pos);

	// -------------------------------------------
	// Backward step for BFECC:
	// From the forward_pos, integrate forward dt again:
	const nanovdb::Vec3f back_velocity = velocitySampler(forward_pos);
	const nanovdb::Vec3f back_pos = pos + back_velocity * scaled_dt;
	const nanovdb::Vec3f value_backward = velocitySampler(back_pos);

	// Error estimation and correction
	const nanovdb::Vec3f error = computeError(velocity, value_backward);
	nanovdb::Vec3f value_corrected = value_forward + error;
	const nanovdb::Vec3f max_correction = computeMaxCorrection(value_forward, velocity);
	value_corrected = clampValue(value_corrected, value_forward - max_correction, value_forward + max_correction);

	// Store the new value
	outVelocity[idx] = value_corrected;
}


void advect_index_grid(HNS::GridIndexedData& data, const float dt,
                       const float voxelSize, const hipStream_t& stream) {
	const size_t totalVoxels = data.size();

	const nanovdb::Vec3f* velocity = reinterpret_cast<nanovdb::Vec3f*>(data.pValues<openvdb::Vec3f>("vel"));
	auto* density = data.pValues<float>("density");
	auto* temperature = data.pValues<float>("temperature");
	auto* fuel = data.pValues<float>("fuel");

	if (!velocity || !density || !temperature || !fuel) {
		throw std::runtime_error("Density data not found in the grid.");
	}

	// Allocate device memory.
	nanovdb::Vec3f* d_velocity = nullptr;
	nanovdb::Coord* d_coords = nullptr;

	float* d_density = nullptr;
	float* d_temperature = nullptr;
	float* d_fuel = nullptr;

	hipMalloc(&d_velocity, totalVoxels * sizeof(nanovdb::Vec3f));
	hipMalloc(&d_coords, totalVoxels * sizeof(nanovdb::Coord));
	hipMalloc(&d_density, totalVoxels * sizeof(float));
	hipMalloc(&d_temperature, totalVoxels * sizeof(float));
	hipMalloc(&d_fuel, totalVoxels * sizeof(float));

	hipMemcpy(d_coords, data.pCoords(), totalVoxels * sizeof(nanovdb::Coord), hipMemcpyHostToDevice);
	hipMemcpy(d_density, density, totalVoxels * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_temperature, temperature, totalVoxels * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_fuel, fuel, totalVoxels * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_velocity, velocity, totalVoxels * sizeof(nanovdb::Vec3f), hipMemcpyHostToDevice);

	// Allocate device memory for the output density.
	float* d_outDensity = nullptr;
	float* d_outTemperature = nullptr;
	float* d_outFuel = nullptr;

	hipMalloc(&d_outDensity, totalVoxels * sizeof(float));
	hipMalloc(&d_outTemperature, totalVoxels * sizeof(float));
	hipMalloc(&d_outFuel, totalVoxels * sizeof(float));

	nanovdb::GridHandle<nanovdb::cuda::DeviceBuffer> handle =
	nanovdb::tools::cuda::voxelsToGrid<nanovdb::ValueOnIndex, nanovdb::Coord*>(d_coords, data.size(), voxelSize);


	const auto gpuGrid = handle.deviceGrid<nanovdb::ValueOnIndex>();

	constexpr int blockSize = 512;
	int numBlocks = (totalVoxels + blockSize - 1) / blockSize;

	advect_idx<<<numBlocks, blockSize, 0, stream>>>(gpuGrid, d_coords, d_velocity, d_density, d_outDensity, totalVoxels, dt, voxelSize);
	advect_idx<<<numBlocks, blockSize, 0, stream>>>(gpuGrid, d_coords, d_velocity, d_temperature, d_outTemperature, totalVoxels, dt, voxelSize);
	advect_idx<<<numBlocks, blockSize, 0, stream>>>(gpuGrid, d_coords, d_velocity, d_fuel, d_outFuel, totalVoxels, dt, voxelSize);

	hipStreamSynchronize(stream);

	hipMemcpy(density, d_outDensity, totalVoxels * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(temperature, d_outTemperature, totalVoxels * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(fuel, d_outFuel, totalVoxels * sizeof(float), hipMemcpyDeviceToHost);

	// Free the allocated device memory.
	hipFree(d_velocity);
	hipFree(d_density);
	hipFree(d_coords);
	hipFree(d_outDensity);
	hipFree(d_temperature);
	hipFree(d_outTemperature);
	hipFree(d_fuel);
	hipFree(d_outFuel);
}


void advect_index_grid_v(HNS::GridIndexedData& data, const float dt,
                       const float voxelSize, const hipStream_t& stream) {
	const size_t totalVoxels = data.size();

	nanovdb::Vec3f* velocity = reinterpret_cast<nanovdb::Vec3f*>(data.pValues<openvdb::Vec3f>("vel"));

	nanovdb::Vec3f* d_velocity = nullptr;
	hipMalloc(&d_velocity, totalVoxels * sizeof(nanovdb::Vec3f));
	// Use synchronous copy for non-pinned memory
	hipMemcpy(d_velocity, velocity, totalVoxels * sizeof(nanovdb::Vec3f), hipMemcpyHostToDevice);

	// Allocate device memory for voxel coordinates.
	nanovdb::Coord* d_coords = nullptr;
	hipMalloc(&d_coords, totalVoxels * sizeof(nanovdb::Coord));
	hipMemcpy(d_coords, data.pCoords(), totalVoxels * sizeof(nanovdb::Coord), hipMemcpyHostToDevice);

	// Allocate device memory for the output density.
	nanovdb::Vec3f* d_outVel = nullptr;
	hipMalloc(&d_outVel, totalVoxels * sizeof(nanovdb::Vec3f));
	hipDeviceSynchronize();

	nanovdb::GridHandle<nanovdb::cuda::DeviceBuffer> handle =
	nanovdb::tools::cuda::voxelsToGrid<nanovdb::ValueOnIndex, nanovdb::Coord*>(d_coords, data.size(), voxelSize);
	const auto gpuGrid = handle.deviceGrid<nanovdb::ValueOnIndex>();
	hipDeviceSynchronize();

	constexpr int blockSize = 256;
	int numBlocks = (totalVoxels + blockSize - 1) / blockSize;
	advect_idx<<<numBlocks, blockSize, 0, stream>>>(gpuGrid, d_coords, d_velocity, d_outVel, totalVoxels, dt, voxelSize);

	// Make sure kernel is finished before copying back
	hipDeviceSynchronize();

	// Use synchronous copy for non-pinned memory
	hipMemcpy(velocity, d_outVel, totalVoxels * sizeof(nanovdb::Vec3f), hipMemcpyDeviceToHost);

	// Free the allocated device memory.
	hipFree(d_velocity);
	hipFree(d_outVel);
	hipFree(d_coords);
}


extern "C" void AdvectIndexGrid(HNS::GridIndexedData& data,
                                const float dt, const float voxelSize, const hipStream_t& stream) {
	advect_index_grid(data, dt, voxelSize, stream);
}

extern "C" void AdvectIndexGridVelocity(HNS::GridIndexedData& data, const float dt,
				const float voxelSize, const hipStream_t& stream) {
	advect_index_grid_v(data, dt, voxelSize, stream);
}
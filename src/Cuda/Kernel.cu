#include "hip/hip_runtime.h"
#pragma once

#include "../Utils/Stencils.hpp"
#include "Kernels.cuh"
#include "nanovdb/tools/cuda/PointsToGrid.cuh"

// Helper function to sample SDF value at a specific coordinate
template <typename Vec3T>
__device__ float sampleSDF(const float* sdfData, const Vec3T& coord, const IndexSampler<float, 1>& sampler) {
	if (!sdfData) return 1.0f;  // If no SDF data, return a value representing "outside" (no collision)
	return sampler(coord);
}


template <typename Vec3T>
__device__ nanovdb::Vec3f gradientSDF(const float* sdfData, const Vec3T& coord, const IndexSampler<float, 1>& sampler,
                                      const float inv_voxelSize) {
	if (!sdfData) return nanovdb::Vec3f(0);  // If no SDF data, return a value representing "outside" (no collision)

	const float right = sampler(coord + Vec3T(1, 0, 0));
	const float left = sampler(coord + Vec3T(-1, 0, 0));
	const float top = sampler(coord + Vec3T(0, 1, 0));
	const float bottom = sampler(coord + Vec3T(0, -1, 0));
	const float front = sampler(coord + Vec3T(0, 0, 1));
	const float back = sampler(coord + Vec3T(0, 0, -1));

	return nanovdb::Vec3f(right - left, top - bottom, front - back) * (0.5f * inv_voxelSize);
}

// Check if a position is inside a collision object
template <typename Vec3T>
__device__ bool isInCollision(const float* sdfData, const Vec3T& pos, const IndexSampler<float, 1>& sampler, float threshold) {
	if (!sdfData) return false;

	const float sdfValue = sampleSDF(sdfData, pos, sampler);
	return sdfValue < threshold;  // Negative inside, positive outside
}

// Compute gradient of SDF at a position to get normal vector
template <typename Vec3T>
__device__ nanovdb::Vec3f getSDFNormal(const float* sdfData, Vec3T& pos, const IndexSampler<float, 1>& sampler, float epsilon) {
	if (!sdfData) return nanovdb::Vec3f(0.0f);

	const nanovdb::Vec3f g = gradientSDF(sdfData, pos, sampler, epsilon);
	const float len = g.length();
	return len > 1e-6f ? g / len : nanovdb::Vec3f(0.0f);
}

__device__ nanovdb::Vec3f applySpecularReflection(const nanovdb::Vec3f& v,
                                                  const nanovdb::Vec3f& n)  // n must be normalised
{
	const float vdotn = v[0] * n[0] + v[1] * n[1] + v[2] * n[2];
	return v - n * (2.0f * vdotn);  // subtract 2×(v·n) n
}

// Apply no-slip boundary condition
__device__ nanovdb::Vec3f applyNoSlipBoundary(const nanovdb::Vec3f& velocity, const nanovdb::Vec3f& normal) {
	// No-slip: remove all velocity at boundary
	// For a point near the boundary, we project the velocity onto the tangent plane
	// and then scale it based on distance to properly enforce no-slip

	// Project velocity onto normal
	const float vdotn = velocity[0] * normal[0] + velocity[1] * normal[1] + velocity[2] * normal[2];

	// Compute the normal component of the velocity
	const nanovdb::Vec3f v_normal = normal * vdotn;

	// Subtract normal component to get tangential component only
	const nanovdb::Vec3f v_tangent = velocity - v_normal;

	// For strict no-slip, even the tangential component should be zero at the boundary
	// We'll return zero velocity (completely stopped by obstacle)
	return v_tangent;  // nanovdb::Vec3f(0.0f);
}

// Kernel to enforce collision boundaries
__global__ void enforceCollisionBoundaries(const nanovdb::NanoGrid<nanovdb::ValueOnIndex>* __restrict__ domainGrid,
                                           const nanovdb::Coord* __restrict__ coords, nanovdb::Vec3f* __restrict__ velocityData,
                                           const float* __restrict__ collisionSDF, const float voxelSize, size_t totalVoxels) {
	const uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= totalVoxels) return;

	if (!collisionSDF) return;

	const IndexOffsetSampler<0> idxSampler(domainGrid);
	const IndexSampler<float, 1> sdfSampler(idxSampler, collisionSDF);
	const nanovdb::Coord coord = coords[idx];

	// Check if we're inside or close to a collision object
	const float sdf_value = sampleSDF(collisionSDF, coord, sdfSampler);

	// Inside collision: set velocity to zero
	if (sdf_value < 0.0f) {
		velocityData[idx] = nanovdb::Vec3f(0.0f);
		return;
	}

	// Within collision margin: apply boundary condition
	const float collisionMargin = 0.1;  // Margin in voxels
	if (sdf_value < collisionMargin) {
		// Get the boundary normal
		const nanovdb::Vec3f normal = getSDFNormal(collisionSDF, coord, sdfSampler, 1.0f / voxelSize);

		// Compute blend factor based on distance (closer to boundary = more damping)
		const float blend = 1.0f - (sdf_value / collisionMargin);

		// Get current velocity
		const nanovdb::Vec3f velocity = velocityData[idx];

		// Apply no-slip boundary condition
		const nanovdb::Vec3f modified_velocity = applyNoSlipBoundary(velocity, normal);

		// Blend between original and modified velocity based on distance to boundary
		velocityData[idx] = velocity * (1.0f - blend) + modified_velocity * blend;
	}
}

__global__ void advect_scalars(const nanovdb::NanoGrid<nanovdb::ValueOnIndex>* __restrict__ domainGrid,
                               const nanovdb::Coord* __restrict__ coords, const nanovdb::Vec3f* __restrict__ velocityData,
                               float** __restrict__ inDataArrays, float** __restrict__ outDataArrays, const int numScalars,
                               const float* __restrict__ collisionSDF, const bool hasCollision, const size_t totalVoxels, const float dt,
                               const float inv_voxelSize) {
	IndexOffsetSampler<0> s_idxSampler(domainGrid);
	const IndexSampler<float, 1> sdfSampler(s_idxSampler, collisionSDF);

	const uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= totalVoxels) return;

	const float scaled_dt = dt * inv_voxelSize;

	const nanovdb::Coord coord = coords[idx];
	uint64_t origIndex = s_idxSampler.offset(coord);
	origIndex = origIndex == 0 ? 0 : origIndex - 1;

	const nanovdb::Vec3f posCell = coord.asVec3s();
	const nanovdb::Vec3f velCenter = velocityData[origIndex];

	// Semi-Lagrangian backtrace
	nanovdb::Vec3f backPos = posCell - velCenter * scaled_dt;

	// Collision handling for backtracing
	if (hasCollision && collisionSDF) {
		// Check if backtraced position is in collision
		if (isInCollision(collisionSDF, backPos, sdfSampler, 0.0f)) {
			// If in collision, do not backtrace through object - use current position instead
			backPos = posCell;
		}
	}

	// Precompute interpolation data for backPos
	struct InterpData {
		uint64_t indices[8];
		float weights[8];
	};
	InterpData backPosData;
	{
		const float x = backPos[0], y = backPos[1], z = backPos[2];
		const int i0 = floor(x), i1 = i0 + 1;
		const int j0 = floor(y), j1 = j0 + 1;
		const int k0 = floor(z), k1 = k0 + 1;
		const float tx = x - i0, ty = y - j0, tz = z - k0;
		const float w000 = (1 - tx) * (1 - ty) * (1 - tz), w100 = tx * (1 - ty) * (1 - tz);
		const float w010 = (1 - tx) * ty * (1 - tz), w110 = tx * ty * (1 - tz);
		const float w001 = (1 - tx) * (1 - ty) * tz, w101 = tx * (1 - ty) * tz;
		const float w011 = (1 - tx) * ty * tz, w111 = tx * ty * tz;

		uint64_t indices[8] = {s_idxSampler.offset(nanovdb::Coord(i0, j0, k0)), s_idxSampler.offset(nanovdb::Coord(i1, j0, k0)),
		                       s_idxSampler.offset(nanovdb::Coord(i0, j1, k0)), s_idxSampler.offset(nanovdb::Coord(i1, j1, k0)),
		                       s_idxSampler.offset(nanovdb::Coord(i0, j0, k1)), s_idxSampler.offset(nanovdb::Coord(i1, j0, k1)),
		                       s_idxSampler.offset(nanovdb::Coord(i0, j1, k1)), s_idxSampler.offset(nanovdb::Coord(i1, j1, k1))};

		for (int j = 0; j < 8; ++j) {
			indices[j] = indices[j] == 0 ? 0 : indices[j] - 1;
		}

		backPosData = {{indices[0], indices[1], indices[2], indices[3], indices[4], indices[5], indices[6], indices[7]},
		               {w000, w100, w010, w110, w001, w101, w011, w111}};
	}

	// Compute velF using precomputed backPosData
	nanovdb::Vec3f velF(0.0f);
#pragma unroll
	for (int j = 0; j < 8; ++j) {
		const nanovdb::Vec3f v = velocityData[backPosData.indices[j]];
		velF += v * backPosData.weights[j];
	}

	nanovdb::Vec3f fwdPos2 = backPos + velF * scaled_dt;

	// Check for collision in the forward trace as well
	if (hasCollision && collisionSDF) {
		if (isInCollision(collisionSDF, fwdPos2, sdfSampler, 0.0f)) {
			fwdPos2 = backPos;
		}
	}

	// Precompute interpolation data for fwdPos2
	InterpData fwdPos2Data;
	{
		const float x = fwdPos2[0], y = fwdPos2[1], z = fwdPos2[2];
		const int i0 = floor(x), i1 = i0 + 1;
		const int j0 = floor(y), j1 = j0 + 1;
		const int k0 = floor(z), k1 = k0 + 1;
		const float tx = x - i0, ty = y - j0, tz = z - k0;
		const float w000 = (1 - tx) * (1 - ty) * (1 - tz), w100 = tx * (1 - ty) * (1 - tz);
		const float w010 = (1 - tx) * ty * (1 - tz), w110 = tx * ty * (1 - tz);
		const float w001 = (1 - tx) * (1 - ty) * tz, w101 = tx * (1 - ty) * tz;
		const float w011 = (1 - tx) * ty * tz, w111 = tx * ty * tz;

		uint64_t indices[8] = {s_idxSampler.offset(nanovdb::Coord(i0, j0, k0)), s_idxSampler.offset(nanovdb::Coord(i1, j0, k0)),
		                       s_idxSampler.offset(nanovdb::Coord(i0, j1, k0)), s_idxSampler.offset(nanovdb::Coord(i1, j1, k0)),
		                       s_idxSampler.offset(nanovdb::Coord(i0, j0, k1)), s_idxSampler.offset(nanovdb::Coord(i1, j0, k1)),
		                       s_idxSampler.offset(nanovdb::Coord(i0, j1, k1)), s_idxSampler.offset(nanovdb::Coord(i1, j1, k1))};

		for (int j = 0; j < 8; ++j) {
			indices[j] = indices[j] == 0 ? 0 : indices[j] - 1;
		}

		fwdPos2Data = {{indices[0], indices[1], indices[2], indices[3], indices[4], indices[5], indices[6], indices[7]},
		               {w000, w100, w010, w110, w001, w101, w011, w111}};
	}

	// Precompute neighbor indices for clamp step
	static __device__ const int offs[6][3] = {{-1, 0, 0}, {+1, 0, 0}, {0, -1, 0}, {0, +1, 0}, {0, 0, -1}, {0, 0, +1}};
	uint32_t nbrIdx[6];
#pragma unroll
	for (int n = 0; n < 6; ++n) {
		nbrIdx[n] = s_idxSampler.offset(coord[0] + offs[n][0], coord[1] + offs[n][1], coord[2] + offs[n][2]);
	}
#pragma unroll
	for (int n = 0; n < 6; ++n) {
		nbrIdx[n] = nbrIdx[n] == 0 ? 0 : nbrIdx[n] - 1;
	}

	// Process each scalar
	for (int s = 0; s < numScalars; ++s) {
		const float* __restrict__ inData = inDataArrays[s];
		float* __restrict__ outData = outDataArrays[s];

		const float phiOrig = inData[origIndex];

		// Compute phiForward with unrolled interpolation
		float phiForward =
		    inData[backPosData.indices[0]] * backPosData.weights[0] + inData[backPosData.indices[1]] * backPosData.weights[1] +
		    inData[backPosData.indices[2]] * backPosData.weights[2] + inData[backPosData.indices[3]] * backPosData.weights[3] +
		    inData[backPosData.indices[4]] * backPosData.weights[4] + inData[backPosData.indices[5]] * backPosData.weights[5] +
		    inData[backPosData.indices[6]] * backPosData.weights[6] + inData[backPosData.indices[7]] * backPosData.weights[7];

		// Compute phiBackward with unrolled interpolation
		float phiBackward =
		    inData[fwdPos2Data.indices[0]] * fwdPos2Data.weights[0] + inData[fwdPos2Data.indices[1]] * fwdPos2Data.weights[1] +
		    inData[fwdPos2Data.indices[2]] * fwdPos2Data.weights[2] + inData[fwdPos2Data.indices[3]] * fwdPos2Data.weights[3] +
		    inData[fwdPos2Data.indices[4]] * fwdPos2Data.weights[4] + inData[fwdPos2Data.indices[5]] * fwdPos2Data.weights[5] +
		    inData[fwdPos2Data.indices[6]] * fwdPos2Data.weights[6] + inData[fwdPos2Data.indices[7]] * fwdPos2Data.weights[7];

		// Correction step
		const float error = phiOrig - phiBackward;
		float phiCorr = phiForward + 0.5f * error;

		// Clamp step
		float minVal = phiOrig, maxVal = phiOrig;
		for (int neighborIndice : nbrIdx) {
			const float val = inData[neighborIndice];
			minVal = fminf(minVal, val);
			maxVal = fmaxf(maxVal, val);
		}
		minVal = fminf(minVal, phiForward);
		maxVal = fmaxf(maxVal, phiForward);
		phiCorr = fmaxf(minVal, fminf(phiCorr, maxVal));

		outData[idx] = phiCorr;
	}
}


__global__ void advect_scalar(const nanovdb::NanoGrid<nanovdb::ValueOnIndex>* __restrict__ domainGrid,
                              const nanovdb::Coord* __restrict__ coords, const nanovdb::Vec3f* __restrict__ velocityData,
                              const float* __restrict__ inData, float* __restrict__ outData, const float* __restrict__ collisionSDF,
                              const bool hasCollision, const size_t totalVoxels, const float dt, const float inv_voxelSize) {
	const uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= totalVoxels) return;

	const float scaled_dt = dt * inv_voxelSize;

	const IndexOffsetSampler<0> idxSampler(domainGrid);
	const auto sdfSampler = IndexSampler<float, 1>(idxSampler, collisionSDF);
	const auto velocitySampler = IndexSampler<nanovdb::Vec3f, 1>(idxSampler, velocityData);
	const auto dataSampler = IndexSampler<float, 1>(idxSampler, inData);

	// The cell coordinate
	const nanovdb::Coord coord = coords[idx];
	const nanovdb::Vec3f posCell = coord.asVec3s();
	const float phiOrig = dataSampler(coord);

	// MAC Velocity for backtrace
	const nanovdb::Vec3f velCenter = velocitySampler(coord);

	// ------------------------------------------------------------------------
	// Forward pass (semi-Lagrangian backtrace)
	// x_forward = x - vel*dt
	nanovdb::Vec3f backPos = posCell - velCenter * scaled_dt;

	// Check for collisions during backtracing
	if (hasCollision && collisionSDF) {
		if (isInCollision(collisionSDF, backPos, sdfSampler, 0.0f)) {
			backPos = posCell;
		}
	}

	const float phiForward = dataSampler(backPos);

	// ------------------------------------------------------------------------
	// Backward pass
	// x_backward = x_forward + u(x_forward)*dt
	// Then compare that to the original value
	const nanovdb::Vec3f velF = velocitySampler(backPos);
	nanovdb::Vec3f fwdPos2 = backPos + velF * scaled_dt;

	// Check for collisions during forward tracing
	if (hasCollision && collisionSDF) {
		if (isInCollision(collisionSDF, fwdPos2, sdfSampler, 0.0f)) {
			fwdPos2 = backPos;
		}
	}

	const float phiBackward = dataSampler(fwdPos2);

	// ------------------------------------------------------------------------
	// Correction
	// error = phiOrig - phiBackward
	// phiCorr = phiForward + 0.5 * error
	const float error = phiOrig - phiBackward;
	float phiCorr = phiForward + 0.5f * error;

	// ------------------------------------------------------------------------
	// Find local min/max in neighborhood for clamping
	float minVal = phiOrig;
	float maxVal = phiOrig;

	// Check 6-neighborhood for min/max values
	for (int dim = 0; dim < 3; ++dim) {
		for (int offset = -1; offset <= 1; offset += 2) {
			nanovdb::Coord neighborCoord = coord;
			neighborCoord[dim] += offset;
			const float neighborVal = dataSampler(neighborCoord);
			minVal = fminf(minVal, neighborVal);
			maxVal = fmaxf(maxVal, neighborVal);
		}
	}

	// Also include the semi-Lagrangian value in min/max computation
	minVal = fminf(minVal, phiForward);
	maxVal = fmaxf(maxVal, phiForward);

	// Clamp the result
	phiCorr = fmaxf(minVal, fminf(phiCorr, maxVal));

	outData[idx] = phiCorr;
}

__global__ void advect_vector(const nanovdb::NanoGrid<nanovdb::ValueOnIndex>* __restrict__ domainGrid,
                              const nanovdb::Coord* __restrict__ coords, const nanovdb::Vec3f* __restrict__ velocityData,
                              nanovdb::Vec3f* __restrict__ outVelocity, const float* __restrict__ collisionSDF, const bool hasCollision,
                              const size_t totalVoxels, const float dt, const float inv_voxelSize) {
	const uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= totalVoxels) return;

	const float scaled_dt = dt * inv_voxelSize;

	const IndexOffsetSampler<0> idxSampler(domainGrid);
	const auto velocitySampler = IndexSampler<nanovdb::Vec3f, 1>(idxSampler, velocityData);
	const auto sdfSampler = IndexSampler<float, 1>(idxSampler, collisionSDF);

	const nanovdb::Coord coord = coords[idx];
	const nanovdb::Vec3f pos = coord.asVec3s();

	// Original velocity at the cell or face
	const nanovdb::Vec3f velOrig = velocitySampler(coord);

	// Forward pass (backtrace)
	nanovdb::Vec3f backPos = pos - velOrig * scaled_dt;

	// Check for collisions during backtracing
	if (hasCollision && collisionSDF) {
		if (isInCollision(collisionSDF, backPos, sdfSampler, 0.0f)) {
			// If we hit a collision, use the original position
			backPos = pos;
		}
	}

	const nanovdb::Vec3f velForward = velocitySampler(backPos);

	// Backward check
	nanovdb::Vec3f fwdPos2 = backPos + velForward * scaled_dt;

	// Check for collisions during forward tracing
	if (hasCollision && collisionSDF) {
		if (isInCollision(collisionSDF, fwdPos2, sdfSampler, 0.0f)) {
			fwdPos2 = backPos;
		}
	}

	const nanovdb::Vec3f velBackward = velocitySampler(fwdPos2);

	// Correction
	const nanovdb::Vec3f errorVec = velOrig - velBackward;
	nanovdb::Vec3f velCorr = velForward + 0.5f * errorVec;

	// Find neighborhood min/max for each component
	nanovdb::Vec3f minVel, maxVel;
	for (int c = 0; c < 3; ++c) {
		minVel[c] = velOrig[c];
		maxVel[c] = velOrig[c];
	}

	// Check 6-neighborhood for min/max values
	for (int dim = 0; dim < 3; ++dim) {
		for (int offset = -1; offset <= 1; offset += 2) {
			nanovdb::Coord neighborCoord = coord;
			neighborCoord[dim] += offset;

			const nanovdb::Vec3f neighborVel = velocitySampler(neighborCoord);
			for (int c = 0; c < 3; ++c) {
				minVel[c] = fminf(minVel[c], neighborVel[c]);
				maxVel[c] = fmaxf(maxVel[c], neighborVel[c]);
			}
		}
	}

	// Also include the semi-Lagrangian value in min/max computation
	for (int c = 0; c < 3; ++c) {
		minVel[c] = fminf(minVel[c], velForward[c]);
		maxVel[c] = fmaxf(maxVel[c], velForward[c]);

		// Clamp the result
		velCorr[c] = fmaxf(minVel[c], fminf(velCorr[c], maxVel[c]));
	}

	// Handle collision boundaries for velocity
	if (hasCollision && collisionSDF) {
		const float sdf_value = sampleSDF(collisionSDF, coord, sdfSampler);

		if (sdf_value < 0.0f) {
			// Inside collision - zero velocity
			velCorr = nanovdb::Vec3f(0.0f);
		} else if (sdf_value < 0.1f) {
			// Near collision - apply no-slip boundary
			const nanovdb::Vec3f normal = getSDFNormal(collisionSDF, coord, sdfSampler, inv_voxelSize);
			const float blend = 1.0f - (sdf_value / 1.5f);

			// Calculate no-slip velocity
			const nanovdb::Vec3f no_slip = applyNoSlipBoundary(velCorr, normal);

			// Blend based on distance
			velCorr = velCorr * (1.0f - blend) + no_slip * blend;
		}
	}

	outVelocity[idx] = velCorr;
}

__global__ void divergence_opt(const nanovdb::NanoGrid<nanovdb::ValueOnIndex>* domainGrid, const nanovdb::Vec3f* velocityData,
                               float* outDivergence, const float inv_dx, const int numLeaves) {
	// Block dimensions matching leaf size
	constexpr int BLOCK_SIZE = 8;

	const int leafIdx = blockIdx.x;

	if (leafIdx >= numLeaves) return;

	const int tidx = threadIdx.x;
	const int tidy = threadIdx.y;
	const int tidz = threadIdx.z;

	const auto& leaf = domainGrid->tree().getFirstNode<0>()[leafIdx];
	const nanovdb::Coord origin = leaf.origin();

	const IndexOffsetSampler<0> idxSampler(domainGrid);
	const auto velocitySampler = IndexSampler<nanovdb::Vec3f, 0>(idxSampler, velocityData);

	// Compute pressure gradient and update velocity
	if (tidx < BLOCK_SIZE && tidy < BLOCK_SIZE && tidz < BLOCK_SIZE) {
		const nanovdb::Coord coord = origin + nanovdb::Coord(tidx, tidy, tidz);

		const nanovdb::Vec3f current = velocitySampler(coord);

		// Average neighboring velocities for each component
		const float xp = 0.5f * (current[0] + velocitySampler(coord + nanovdb::Coord(1, 0, 0))[0]);
		const float xm = 0.5f * (current[0] + velocitySampler(coord - nanovdb::Coord(1, 0, 0))[0]);

		const float yp = 0.5f * (current[1] + velocitySampler(coord + nanovdb::Coord(0, 1, 0))[1]);
		const float ym = 0.5f * (current[1] + velocitySampler(coord - nanovdb::Coord(0, 1, 0))[1]);

		const float zp = 0.5f * (current[2] + velocitySampler(coord + nanovdb::Coord(0, 0, 1))[2]);
		const float zm = 0.5f * (current[2] + velocitySampler(coord - nanovdb::Coord(0, 0, 1))[2]);

		const float divergence = (xp - xm + yp - ym + zp - zm) * inv_dx;

		const auto idx = idxSampler.offset(coord);
		const auto cidx = idx == 0 ? 0 : idx - 1;
		outDivergence[cidx] = divergence;
	}
}


__global__ void divergence(const nanovdb::NanoGrid<nanovdb::ValueOnIndex>* __restrict__ domainGrid,
                           const nanovdb::Coord* __restrict__ d_coord, const nanovdb::Vec3f* __restrict__ velocityData,
                           float* __restrict__ outDivergence, const float inv_dx, const size_t totalVoxels) {
	const size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= totalVoxels) return;

	const nanovdb::Vec3f center = velocityData[tid];

	const nanovdb::Coord c = d_coord[tid];
	const IndexOffsetSampler<0> idxSampler(domainGrid);
	const auto velocitySampler = IndexSampler<nanovdb::Vec3f, 0>(idxSampler, velocityData);

	const float xp = (center + velocitySampler(c + nanovdb::Coord(1, 0, 0)))[0] * 0.5f;
	const float xm = (center + velocitySampler(c - nanovdb::Coord(1, 0, 0)))[0] * 0.5f;
	const float yp = (center + velocitySampler(c + nanovdb::Coord(0, 1, 0)))[1] * 0.5f;
	const float ym = (center + velocitySampler(c - nanovdb::Coord(0, 1, 0)))[1] * 0.5f;
	const float zp = (center + velocitySampler(c + nanovdb::Coord(0, 0, 1)))[2] * 0.5f;
	const float zm = (center + velocitySampler(c - nanovdb::Coord(0, 0, 1)))[2] * 0.5f;

	outDivergence[tid] = (xp - xm + yp - ym + zp - zm) * inv_dx;
}

__global__ void redBlackGaussSeidelUpdate_opt(const nanovdb::NanoGrid<nanovdb::ValueOnIndex>* domainGrid, const float* divergence,
                                              float* pressure, const float dx, const size_t totalVoxels, const int color,
                                              const float omega) {
	constexpr int BLOCK_SIZE = 8;
	__shared__ float s_pressure[BLOCK_SIZE + 2][BLOCK_SIZE + 2][BLOCK_SIZE + 2];

	const int leafIdx = blockIdx.x;
	const int tidx = threadIdx.x;
	const int tidy = threadIdx.y;
	const int tidz = threadIdx.z;

	if (leafIdx >= domainGrid->tree().nodeCount(0)) return;

	const auto& leaf = domainGrid->tree().getFirstNode<0>()[leafIdx];
	const nanovdb::Coord origin = leaf.origin();
	const IndexOffsetSampler<0> idxSampler(domainGrid);
	const auto pSampler = IndexSampler<float, 0>(idxSampler, pressure);

	// Load pressure halo (+1 in all directions)
	for (int idx = tidz * BLOCK_SIZE * BLOCK_SIZE + tidy * BLOCK_SIZE + tidx; idx < (BLOCK_SIZE + 2) * (BLOCK_SIZE + 2) * (BLOCK_SIZE + 2);
	     idx += BLOCK_SIZE * BLOCK_SIZE * BLOCK_SIZE) {
		const int dz = idx / ((BLOCK_SIZE + 2) * (BLOCK_SIZE + 2));
		const int dy = (idx % ((BLOCK_SIZE + 2) * (BLOCK_SIZE + 2))) / (BLOCK_SIZE + 2);
		const int dx = idx % (BLOCK_SIZE + 2);

		const nanovdb::Coord coord(origin.x() + dx - 1, origin.y() + dy - 1, origin.z() + dz - 1);
		s_pressure[dz][dy][dx] = pSampler(coord);
	}

	__syncthreads();

	// Process center region
	if (tidx < BLOCK_SIZE && tidy < BLOCK_SIZE && tidz < BLOCK_SIZE) {
		const nanovdb::Coord coord = origin + nanovdb::Coord(tidx, tidy, tidz);
		const int i = coord.x(), j = coord.y(), k = coord.z();

		// Red/black check
		if (((i + j + k) & 1) != color) return;

		// Find linear index in d_coord array
		size_t tid = idxSampler.offset(coord);
		tid = tid == 0 ? 0 : tid - 1;

		if (tid >= totalVoxels) return;

		// Shared memory indices with halo offset
		const int lx = tidx + 1;
		const int ly = tidy + 1;
		const int lz = tidz + 1;

		// Stencil accesses from shared memory
		const float pxp1 = s_pressure[lz][ly][lx + 1];
		const float pxm1 = s_pressure[lz][ly][lx - 1];
		const float pyp1 = s_pressure[lz][ly + 1][lx];
		const float pym1 = s_pressure[lz][ly - 1][lx];
		const float pzp1 = s_pressure[lz + 1][ly][lx];
		const float pzm1 = s_pressure[lz - 1][ly][lx];

		// Gauss-Seidel update
		const float dx2 = dx * dx;
		constexpr float inv6 = 0.166666667f;
		const float divVal = divergence[tid];
		const float pOld = s_pressure[lz][ly][lx];  // From shared memory

		const float pGS = ((pxp1 + pxm1 + pyp1 + pym1 + pzp1 + pzm1) - divVal * dx2) * inv6;
		pressure[tid] = pOld + omega * (pGS - pOld);
	}
}


__global__ void redBlackGaussSeidelUpdate(const nanovdb::NanoGrid<nanovdb::ValueOnIndex>* __restrict__ domainGrid,
                                          const nanovdb::Coord* __restrict__ d_coord, const float* __restrict__ divergence,
                                          float* __restrict__ pressure, const float dx, const size_t totalVoxels, const int color,
                                          const float omega) {
	const size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= totalVoxels) return;

	const nanovdb::Coord c = d_coord[tid];
	const int i = c.x(), j = c.y(), k = c.z();

	// Skip if wrong color
	if (((i + j + k) & 1) != color) return;

	const IndexOffsetSampler<0> idxSampler(domainGrid);
	const auto pSampler = IndexSampler<float, 0>(idxSampler, pressure);

	// Pre-compute common factors
	const float dx2 = dx * dx;
	constexpr float inv6 = 0.166666667;

	const float pxp1 = pSampler(nanovdb::Coord(i + 1, j, k));
	const float pxm1 = pSampler(nanovdb::Coord(i - 1, j, k));
	const float pyp1 = pSampler(nanovdb::Coord(i, j + 1, k));
	const float pym1 = pSampler(nanovdb::Coord(i, j - 1, k));
	const float pzp1 = pSampler(nanovdb::Coord(i, j, k + 1));
	const float pzm1 = pSampler(nanovdb::Coord(i, j, k - 1));

	const float divVal = divergence[tid];
	const float pOld = pressure[tid];

	const float pGS = ((pxp1 + pxm1 + pyp1 + pym1 + pzp1 + pzm1) - divVal * dx2) * inv6;
	pressure[tid] = pOld + omega * (pGS - pOld);
}

__global__ void restrict_to_4x4x4(const float* inData, float* outData, const size_t totalVoxels) {
	// totalVoxels should be 64 (for a 4x4x4 coarse grid)
	const size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= totalVoxels) return;

	// Map tid [0,63] to coarse grid coordinates in a 4x4x4 block:
	const int ic = tid % 4;
	const int jc = (tid / 4) % 4;
	const int kc = tid / 16;

	// Each coarse cell covers a 2x2x2 block in the fine grid.
	// Fine grid is 8x8x8; compute starting indices for the corresponding block.
	const int i_fine_start = ic * 2;
	const int j_fine_start = jc * 2;
	const int k_fine_start = kc * 2;

	float sum = 0.0f;
	// Loop over the 2x2x2 block
	for (int dz = 0; dz < 2; ++dz) {
		for (int dy = 0; dy < 2; ++dy) {
			for (int dx = 0; dx < 2; ++dx) {
				int i_fine = i_fine_start + dx;
				int j_fine = j_fine_start + dy;
				int k_fine = k_fine_start + dz;
				// Assuming fine grid is stored in x-fastest order:
				// Index = i + j * (8) + k * (8*8) where grid dimensions are 8×8×8.
				const int index = i_fine + j_fine * 8 + k_fine * 64;
				sum += inData[index];
			}
		}
	}
	// Average the sum of 8 fine cells
	outData[tid] = sum / 8.0f;
}


__global__ void redBlackGaussSeidelUpdate_single(const IndexOffsetSampler<0>& sampler, const nanovdb::Coord* d_coord,
                                                 const float* divergence, float* pressure, const float dx, const size_t totalVoxels,
                                                 const float omega, const int color) {
	const size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= totalVoxels) return;

	const nanovdb::Coord c = d_coord[tid];
	const int i = c.x(), j = c.y(), k = c.z();

	// Skip if wrong color
	if (((i + j + k) & 1) != color) return;

	const auto pSampler = IndexSampler<float, 0>(sampler, pressure);

	// Pre-compute common factors
	const float dx2 = dx * dx;
	constexpr float inv6 = 0.166666667;

	const float pxp1 = pSampler(nanovdb::Coord(i + 1, j, k));
	const float pxm1 = pSampler(nanovdb::Coord(i - 1, j, k));
	const float pyp1 = pSampler(nanovdb::Coord(i, j + 1, k));
	const float pym1 = pSampler(nanovdb::Coord(i, j - 1, k));
	const float pzp1 = pSampler(nanovdb::Coord(i, j, k + 1));
	const float pzm1 = pSampler(nanovdb::Coord(i, j, k - 1));

	const float divVal = divergence[tid];
	const float pOld = pressure[tid];

	const float pGS = ((pxp1 + pxm1 + pyp1 + pym1 + pzp1 + pzm1) - divVal * dx2) * inv6;
	pressure[tid] = pOld + omega * (pGS - pOld);
}


__global__ void subtractPressureGradient_opt(const nanovdb::NanoGrid<nanovdb::ValueOnIndex>* __restrict__ domainGrid,
                                             const nanovdb::Vec3f* __restrict__ velocity, const float* __restrict__ pressure,
                                             nanovdb::Vec3f* __restrict__ out, const float inv_voxelSize, const size_t numLeaves) {
	constexpr int BLOCK_SIZE = 8;
	__shared__ float s_pressure[BLOCK_SIZE + 2][BLOCK_SIZE + 2][BLOCK_SIZE + 2];

	const int leafIdx = blockIdx.x;
	const int tidx = threadIdx.x;
	const int tidy = threadIdx.y;
	const int tidz = threadIdx.z;

	if (leafIdx >= numLeaves) return;

	const auto& leaf = domainGrid->tree().getFirstNode<0>()[leafIdx];
	const nanovdb::Coord origin = leaf.origin();

	const IndexOffsetSampler<0> idxSampler(domainGrid);
	const auto velocitySampler = IndexSampler<nanovdb::Vec3f, 0>(idxSampler, velocity);
	const auto pressureSampler = IndexSampler<float, 0>(idxSampler, pressure);

	for (int idx = tidz * BLOCK_SIZE * BLOCK_SIZE + tidy * BLOCK_SIZE + tidx; idx < (BLOCK_SIZE + 2) * (BLOCK_SIZE + 2) * (BLOCK_SIZE + 2);
	     idx += BLOCK_SIZE * BLOCK_SIZE * BLOCK_SIZE) {
		const int dz = idx / ((BLOCK_SIZE + 2) * (BLOCK_SIZE + 2));
		const int dy = (idx % ((BLOCK_SIZE + 2) * (BLOCK_SIZE + 2))) / (BLOCK_SIZE + 2);
		const int dx = idx % (BLOCK_SIZE + 2);

		const nanovdb::Coord coord(origin.x() + dx - 1, origin.y() + dy - 1, origin.z() + dz - 1);
		s_pressure[dz][dy][dx] = pressureSampler(coord);
	}

	__syncthreads();

	if (tidx < BLOCK_SIZE && tidy < BLOCK_SIZE && tidz < BLOCK_SIZE) {
		const int lx = tidx + 1;
		const int ly = tidy + 1;
		const int lz = tidz + 1;

		const nanovdb::Coord coord = origin + nanovdb::Coord(tidx, tidy, tidz);

		// Load current cell's
		const nanovdb::Vec3f& u_star_c = velocitySampler(coord);

		// Pressure values at neighbours
		const float p_xp = s_pressure[lz][ly][lx + 1];  // p(i+1, j, k)
		const float p_xm = s_pressure[lz][ly][lx - 1];  // p(i-1, j, k)
		const float p_yp = s_pressure[lz][ly + 1][lx];  // p(i, j+1, k)
		const float p_ym = s_pressure[lz][ly - 1][lx];  // p(i, j-1, k)
		const float p_zp = s_pressure[lz + 1][ly][lx];  // p(i, j, k+1)
		const float p_zm = s_pressure[lz - 1][ly][lx];  // p(i, j, k-1)

		// Central difference gradient: grad(p)_x = (p(i+1) - p(i-1)) / (2*dx)
		// Multiply by 0.5f * inv_voxelSize which is 1 / (2 * dx)
		const float gradP_x = (p_xp - p_xm) * 0.5f * inv_voxelSize;
		const float gradP_y = (p_yp - p_ym) * 0.5f * inv_voxelSize;
		const float gradP_z = (p_zp - p_zm) * 0.5f * inv_voxelSize;

		// Form the gradient vector at the cell center
		const nanovdb::Vec3f gradP_c = {gradP_x, gradP_y, gradP_z};

		// --- Apply Pressure Gradient ---
		// u_n+1 = u* - dt * grad(p)  (Assuming rho=1)
		// Apply the update using the time step dt
		const nanovdb::Vec3f u_final_c = u_star_c - gradP_c;

		auto idx = idxSampler.offset(coord);
		idx = idx == 0 ? 0 : idx - 1;
		out[idx] = u_final_c;
	}
}


__global__ void subtractPressureGradient(const nanovdb::NanoGrid<nanovdb::ValueOnIndex>* domainGrid, const nanovdb::Coord* d_coords,
                                         const size_t totalVoxels, const nanovdb::Vec3f* velocity, const float* pressure,
                                         nanovdb::Vec3f* out, const float* __restrict__ collisionSDF, const bool hasCollision,
                                         const float inv_voxelSize) {
	const size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= totalVoxels) return;

	// Samplers
	const IndexOffsetSampler<0> idxSampler(domainGrid);
	const auto pressureSampler = IndexSampler<float, 0>(idxSampler, pressure);
	const auto sdfSampler = IndexSampler<float, 1>(idxSampler, collisionSDF);

	// The cell center coordinate associated with this thread
	const nanovdb::Coord c = d_coords[tid];
	const nanovdb::Vec3f pos = c.asVec3s();

	// Get the intermediate cell-centered velocity u*
	const nanovdb::Vec3f u_star_c = velocity[tid];

	// --- Calculate Pressure Gradient at Cell Center (i, j, k) using Central Differences ---

	// Pressure values at neighbours
	const float p_xp = pressureSampler(c + nanovdb::Coord(1, 0, 0));  // p(i+1, j, k)
	const float p_xm = pressureSampler(c - nanovdb::Coord(1, 0, 0));  // p(i-1, j, k)
	const float p_yp = pressureSampler(c + nanovdb::Coord(0, 1, 0));  // p(i, j+1, k)
	const float p_ym = pressureSampler(c - nanovdb::Coord(0, 1, 0));  // p(i, j-1, k)
	const float p_zp = pressureSampler(c + nanovdb::Coord(0, 0, 1));  // p(i, j, k+1)
	const float p_zm = pressureSampler(c - nanovdb::Coord(0, 0, 1));  // p(i, j, k-1)

	// Central difference gradient: grad(p)_x = (p(i+1) - p(i-1)) / (2*dx)
	// Multiply by 0.5f * inv_voxelSize which is 1 / (2 * dx)
	const float gradP_x = (p_xp - p_xm);
	const float gradP_y = (p_yp - p_ym);
	const float gradP_z = (p_zp - p_zm);

	// Form the gradient vector at the cell center
	const nanovdb::Vec3f gradP_c = nanovdb::Vec3f(gradP_x, gradP_y, gradP_z) * 0.5f * inv_voxelSize;

	// --- Apply Pressure Gradient ---
	// u_n+1 = u* - dt * grad(p)
	// Apply the update using the time step dt
	nanovdb::Vec3f u_final_c = u_star_c - gradP_c;

	// Handle collision boundaries
	if (hasCollision && collisionSDF) {
		const float sdf_value = sampleSDF(collisionSDF, c, sdfSampler);

		if (sdf_value < 0.0f) {
			// Inside collision - zero velocity
			u_final_c = nanovdb::Vec3f(0.0f);
		} else if (sdf_value < 0.1f) {
			// Near collision - apply no-slip boundary
			const nanovdb::Vec3f normal = getSDFNormal(collisionSDF, c, sdfSampler, inv_voxelSize);
			const float blend = 1.0f - (sdf_value / 0.1f);

			// Calculate no-slip velocity
			const nanovdb::Vec3f no_slip = applyNoSlipBoundary(u_final_c, normal);

			// Blend based on distance
			u_final_c = u_final_c * (1.0f - blend) + no_slip * blend;
		}
	}

	out[tid] = u_final_c;
}

__global__ void temperature_buoyancy(const nanovdb::Vec3f* velocityData, const float* tempData, nanovdb::Vec3f* outVel, const float dt,
                                     const float ambient_temp, const float buoyancy_strength, const size_t totalVoxels) {
	const uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= totalVoxels) return;

	const nanovdb::Vec3f vel = velocityData[idx];
	const float temp = tempData[idx];
	if (temp <= ambient_temp) {
		outVel[idx] = vel;
		return;
	}

	const float tempDiff = temp - ambient_temp;
	const nanovdb::Vec3f buoyancyForce(0.0f, fmaxf(0.0f, tempDiff * buoyancy_strength), 0.0f);

	outVel[idx] = vel + buoyancyForce * dt;
}

__global__ void combustion(const nanovdb::NanoGrid<nanovdb::ValueOnIndex>* domainGrid, const nanovdb::Coord* __restrict__ d_coords,
                           const float* __restrict__ fuelData, const float* __restrict__ tempData, float* __restrict__ outFuel,
                           float* __restrict__ outTemp, const float dt, float ignition_temp, float combustion_rate, float heat_release,
                           size_t totalVoxels) {
	const uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= totalVoxels) return;


	const float fuel = fuelData[idx];
	const float temp = tempData[idx];
	float newFuel = fuel;
	float newTemp = temp;

	if (fuel > 0.0f && temp >= ignition_temp) {
		const float fuelBurned = fminf(fuel, combustion_rate * dt);
		newFuel -= fuelBurned;
		newTemp += fuelBurned * heat_release;
	}

	outFuel[idx] = newFuel;
	outTemp[idx] = newTemp;
}


__global__ void diffusion(const nanovdb::NanoGrid<nanovdb::ValueOnIndex>* domainGrid, const nanovdb::Coord* __restrict__ d_coords,
                          const float* tempData, const float* fuelData, float* outTemp, float* outFuel, const float dt, float temp_diff,
                          float fuel_diff, float ambient_temp, size_t totalVoxels) {
	const uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= totalVoxels) return;

	const IndexOffsetSampler<0> sampler(domainGrid);
	const auto tempSampler = IndexSampler<float, 0>(sampler, tempData);
	const auto fuelSampler = IndexSampler<float, 0>(sampler, fuelData);

	const nanovdb::Coord coord = d_coords[idx];
	const float centerTemp = tempSampler(coord);
	const float centerFuel = fuelSampler(coord);
	float tempLaplacian = 0.0f;
	float fuelLaplacian = 0.0f;
	int neighbors = 0;

	// Check each of the 6 direct neighbors
	nanovdb::Coord offsets[6] = {{1, 0, 0}, {-1, 0, 0}, {0, 1, 0}, {0, -1, 0}, {0, 0, 1}, {0, 0, -1}};

	for (auto offset : offsets) {
		const nanovdb::Coord neighborCoord = coord + offset;

		// Get the neighbor's index and values
		const float neighborTemp = tempSampler(neighborCoord);
		if (neighborTemp == 0.0f) continue;  // Skip inactive voxels

		const float neighborFuel = fuelSampler(neighborCoord);
		if (neighborFuel == 0.0f) continue;  // Skip inactive voxels

		// Accumulate Laplacian (difference from center to neighbor)
		tempLaplacian += (neighborTemp - centerTemp);
		fuelLaplacian += (neighborFuel - centerFuel);
		neighbors++;
	}

	// Apply diffusion if we have valid neighbors
	if (neighbors > 0) {
		outTemp[idx] = centerTemp + temp_diff * dt * tempLaplacian;
		outFuel[idx] = centerFuel + fuel_diff * dt * fuelLaplacian;
	} else {
		outTemp[idx] = centerTemp;
		outFuel[idx] = centerFuel;
	}

	// Apply cooling effect
	outTemp[idx] += (ambient_temp - outTemp[idx]) * dt * 0.1f;
}


__global__ void combustion_oxygen(const float* fuelData, const float* wasteData, const float* temperatureData, float* divergenceData,
                                  const float* flameData, float* outFuel, float* outWaste, float* outTemperature, float* outFlame,
                                  const float temp_gain, const float expansion, size_t totalVoxels) {
	const uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= totalVoxels) return;

	// Load input values for the current voxel
	float fuel = fuelData[idx];
	float waste = wasteData[idx];
	float temperature = temperatureData[idx];
	float flame = flameData[idx];

	// Apply fuel threshold
	if (fuel < 0.001f) {
		fuel = 0.0f;
	}

	// Calculate available oxygen
	float oxygen = 1.0f - fuel - waste;
	if (oxygen < 0.0f) {
		// Invalid state; copy inputs to outputs
		outFuel[idx] = fuel;
		outWaste[idx] = waste;
		outTemperature[idx] = temperature;
		outFlame[idx] = flame;
		return;
	}

	// Calculate burn amount (oxygen-limited, scaled by ratio)
	float burn = fminf(oxygen, fuel);

	// Update fields
	float newFuel = fuel - burn;
	float newWaste = waste + burn * 2.0f;                      // Fuel + oxygen consumed
	float newFlame = fmaxf(flame, fminf(1.0f, burn * 10.0f));  // Flame intensity
	float newTemperature = temperature + burn * temp_gain;

	// Write updated values to output arrays
	outFuel[idx] = newFuel;
	outWaste[idx] = newWaste;
	outTemperature[idx] = newTemperature;
	divergenceData[idx] += burn * expansion;
	outFlame[idx] = newFlame;
}


// Vorticity Confinement Kernel
__global__ void vorticityConfinement(const nanovdb::NanoGrid<nanovdb::ValueOnIndex>* __restrict__ domainGrid,
                                     const nanovdb::Coord* __restrict__ d_coord, const nanovdb::Vec3f* __restrict__ velocityData,
                                     nanovdb::Vec3f* __restrict__ outForce, const float dt, const float inv_dx,
                                     const float confinementScale, const float factorScale, const size_t totalVoxels) {
	const size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= totalVoxels) return;

	const nanovdb::Coord c = d_coord[tid];

	// Initialize sampler with boundary checks
	const IndexOffsetSampler<0> idxSampler(domainGrid);
	const auto velocitySampler = IndexSampler<nanovdb::Vec3f, 0>(idxSampler, velocityData);
	const float factor = 0.5 * inv_dx;

	// Compute current cell's vorticity vector
	const nanovdb::Vec3f u_pX = velocitySampler(c + nanovdb::Coord(1, 0, 0));
	const nanovdb::Vec3f u_mX = velocitySampler(c - nanovdb::Coord(1, 0, 0));
	const nanovdb::Vec3f u_pY = velocitySampler(c + nanovdb::Coord(0, 1, 0));
	const nanovdb::Vec3f u_mY = velocitySampler(c - nanovdb::Coord(0, 1, 0));
	const nanovdb::Vec3f u_pZ = velocitySampler(c + nanovdb::Coord(0, 0, 1));
	const nanovdb::Vec3f u_mZ = velocitySampler(c - nanovdb::Coord(0, 0, 1));

	const float omega_x = ((u_pY[2] - u_mY[2]) - (u_pZ[1] - u_mZ[1])) * factor;
	const float omega_y = ((u_pZ[0] - u_mZ[0]) - (u_pX[2] - u_mX[2])) * factor;
	const float omega_z = ((u_pX[1] - u_mX[1]) - (u_pY[0] - u_mY[0])) * factor;


	// X-direction neighbors
	const float vortMag_pX = computeVorticityMag(velocitySampler, c + nanovdb::Coord(factorScale, 0, 0), factor);
	const float vortMag_mX = computeVorticityMag(velocitySampler, c - nanovdb::Coord(factorScale, 0, 0), factor);

	// Y-direction neighbors
	const float vortMag_pY = computeVorticityMag(velocitySampler, c + nanovdb::Coord(0, factorScale, 0), factor);
	const float vortMag_mY = computeVorticityMag(velocitySampler, c - nanovdb::Coord(0, factorScale, 0), factor);

	// Z-direction neighbors
	const float vortMag_pZ = computeVorticityMag(velocitySampler, c + nanovdb::Coord(0, 0, factorScale), factor);
	const float vortMag_mZ = computeVorticityMag(velocitySampler, c - nanovdb::Coord(0, 0, factorScale), factor);

	// Compute gradient with safe differences
	const float grad_x = (vortMag_pX - vortMag_mX) * 0.5f * inv_dx;
	const float grad_y = (vortMag_pY - vortMag_mY) * 0.5f * inv_dx;
	const float grad_z = (vortMag_pZ - vortMag_mZ) * 0.5f * inv_dx;

	// Normalize gradient vector
	const float gradLen = sqrtf(grad_x * grad_x + grad_y * grad_y + grad_z * grad_z) + 1e-5f;
	const float Nx = grad_x / gradLen;
	const float Ny = grad_y / gradLen;
	const float Nz = grad_z / gradLen;

	// Compute confinement force using cross product
	outForce[tid] = velocityData[tid] + nanovdb::Vec3f{confinementScale * (Ny * omega_z - Nz * omega_y),
	                                                   confinementScale * (Nz * omega_x - Nx * omega_z),
	                                                   confinementScale * (Nx * omega_y - Ny * omega_x)} *
	                                        dt;
}
#include "hip/hip_runtime.h"
#include <nanovdb/NanoVDB.h>
#include <nanovdb/util/SampleFromVoxels.h>

#include <cuda/std/cmath>
#include <cuda/std/__algorithm/clamp.h>

#include "utils.cuh"

extern "C" void vel_thrust_kernel(nanovdb::Vec3fGrid* deviceGrid, const nanovdb::Vec3fGrid* velGrid,
                                  const uint64_t leafCount, const float voxelSize, const float dt) {
	constexpr unsigned int numThreads = 256;
	const unsigned int numVoxels = 512 * leafCount;
	const unsigned int numBlocks = blocksPerGrid(numVoxels, numThreads);

	lambdaKernel<<<numBlocks, numThreads>>>(numVoxels, [deviceGrid, velGrid, voxelSize,
	                                                    dt] __device__(const uint64_t n) {
		auto& dtree = deviceGrid->tree();
		const auto& vtree = velGrid->tree();

		auto* leaf_d = dtree.getFirstNode<0>() + (n >> 9);
		const int i_d = n & 511;

		const auto* leaf_v = vtree.getFirstNode<0>() + (n >> 9);

		const auto velAccessor = velGrid->getAccessor();
		const auto velSampler = nanovdb::createSampler<1>(velAccessor);

		if (leaf_v->isActive()) {
			// Get the position of the voxel in index space
			const nanovdb::Coord voxelCoord = leaf_v->offsetToGlobalCoord(i_d);
			const nanovdb::Vec3f voxelCoordf = voxelCoord.asVec3s();
			const nanovdb::Vec3f velocity = velSampler(voxelCoordf);

			// Forward step
			const nanovdb::Vec3f forward_pos = voxelCoordf - velocity * (dt / voxelSize);
			const nanovdb::Vec3f v_forward = velSampler(forward_pos);

			// Backward step
			const nanovdb::Vec3f back_pos = voxelCoordf + velSampler(forward_pos) * (dt / voxelSize);
			const nanovdb::Vec3f v_backward = velSampler(back_pos);

			// Error estimation and correction
			const nanovdb::Vec3f error = 0.5f * (velocity - v_backward);
			nanovdb::Vec3f v_corrected = v_forward + error;

			// Limit the correction based on the neighborhood of the forward position
			const auto max_correction = nanovdb::Vec3f(cuda::std::abs(0.5f * (v_forward[0] - velocity[0])),
			                                           cuda::std::abs(0.5f * (v_forward[1] - velocity[1])),
			                                           cuda::std::abs(0.5f * (v_forward[2] - velocity[2])));
			v_corrected[0] =
			    cuda::std::clamp(v_corrected[0], v_forward[0] - max_correction[0], v_forward[0] + max_correction[0]);
			v_corrected[1] =
			    cuda::std::clamp(v_corrected[1], v_forward[1] - max_correction[1], v_forward[1] + max_correction[1]);
			v_corrected[2] =
			    cuda::std::clamp(v_corrected[2], v_forward[2] - max_correction[2], v_forward[2] + max_correction[2]);

			// Final advection (blend between semi-Lagrangian and BFECC result)
			constexpr float blend_factor = 0.8f;  // Adjust this value between 0 and 1
			nanovdb::Vec3f new_velocity;
			new_velocity[0] = lerp(v_forward[0], v_corrected[0], blend_factor);
			new_velocity[1] = lerp(v_forward[1], v_corrected[1], blend_factor);
			new_velocity[2] = lerp(v_forward[2], v_corrected[2], blend_factor);

			// Set the new velocity value
			leaf_d->setValue(voxelCoord, new_velocity);
		}
	});
}

extern "C" void thrust_kernel(nanovdb::FloatGrid* deviceGrid, const nanovdb::Vec3fGrid* velGrid, const int leafCount,
                              const float voxelSize, const float dt) {
	constexpr unsigned int numThreads = 256;
	const unsigned int numVoxels = 512 * leafCount;
	const unsigned int numBlocks = blocksPerGrid(numVoxels, numThreads);


	// TODO: Race condition Read-Write on deviceGrid
	// Somehow make a deep copy to have a readDeviceGrid and writeDeviceGrid
	lambdaKernel<<<numBlocks, numThreads>>>(numVoxels, [deviceGrid, velGrid, voxelSize, dt] __device__(const size_t n) {
		auto& dtree = deviceGrid->tree();
		auto& vtree = velGrid->tree();

		auto* leaf_d = dtree.getFirstNode<0>() + (n >> 9);
		const int i_d = n & 511;

		auto* leaf_v = vtree.getFirstNode<0>() + (n >> 9);

		const auto velAccessor = velGrid->getAccessor();
		const auto denAccessor = deviceGrid->getAccessor();
		const auto velSampler = nanovdb::createSampler<1>(velAccessor);
		const auto denSampler = nanovdb::createSampler<1>(denAccessor);

		if (leaf_v->isActive()) {
			// Get the position of the voxel in index space
			const nanovdb::Coord voxelCoord = leaf_d->offsetToGlobalCoord(i_d);
			const nanovdb::Vec3f voxelCoordf = voxelCoord.asVec3s();
			const float density = denSampler(voxelCoordf);

			// Forward step
			const nanovdb::Vec3f forward_pos = voxelCoordf - velSampler(voxelCoordf) * (dt / voxelSize);
			const float d_forward = denSampler(forward_pos);

			// Backward step
			const nanovdb::Vec3f back_pos = voxelCoordf + velSampler(forward_pos) * (dt / voxelSize);
			const float d_backward = denSampler(back_pos);

			// Error estimation and correction
			const float error = 0.5f * (density - d_backward);
			float d_corrected = d_forward + error;

			// Limit the correction based on the neighborhood of the forward position
			const float max_correction = 0.5f * cuda::std::fabs(d_forward - density);
			d_corrected = cuda::std::clamp(d_corrected, d_forward - max_correction, d_forward + max_correction);

			// Final advection (blend between semi-Lagrangian and BFECC result)
			constexpr float blend_factor = 0.8f;  // Adjust this value between 0 and 1
			float new_density = lerp(d_forward, d_corrected, blend_factor);

			// Ensure non-negativity
			new_density = cuda::std::fmax(0.0f, new_density);

			// Set the new density value
			leaf_d->setValue(voxelCoord, new_density);
		}
	});
	cudaCheckError();
}

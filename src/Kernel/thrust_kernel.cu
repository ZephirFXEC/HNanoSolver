#include "hip/hip_runtime.h"
#include <cuda/std/__algorithm/clamp.h>
#include <nanovdb/NanoVDB.h>
#include <nanovdb/util/GridBuilder.h>
#include <nanovdb/util/SampleFromVoxels.h>

#include <cuda/std/cmath>

#include "utils.cuh"


extern "C" void get_pos_val(nanovdb::FloatGrid* grid, const size_t leafCount, hipStream_t stream, nanovdb::Coord* h_coords, float* h_values, size_t& count) {

	size_t* voxelCount = nullptr;
	cudaCheck(hipMalloc(&voxelCount, sizeof(size_t)));
	cudaCheck(hipMemset(voxelCount, 0, sizeof(size_t)));

	constexpr unsigned int numThreads = 256;
	const unsigned int numVoxels = 512 * leafCount;
	const unsigned int numBlocks = blocksPerGrid(numVoxels, numThreads);

	nanovdb::Coord* d_coords = nullptr;
	float* d_values = nullptr;

	cudaCheck(hipMalloc(&d_coords, numVoxels * sizeof(nanovdb::Coord)));
	cudaCheck(hipMalloc(&d_values, numVoxels * sizeof(float)));
	cudaCheck(hipMemset(d_coords, 0, numVoxels * sizeof(nanovdb::Coord)));
	cudaCheck(hipMemset(d_values, 0, numVoxels * sizeof(float)));

	lambdaKernel<<<numBlocks, numThreads, 0, stream>>>(numVoxels, [grid, voxelCount, d_coords, d_values] __device__(const size_t n) {
		auto& tree = grid->tree();
		const auto* leaf = tree.getFirstNode<0>() + (n >> 9);
		const auto acc = tree.getAccessor();
		const int i = n & 511;
		if (leaf->isActive()) {
			const nanovdb::Coord voxelCoord = leaf->offsetToGlobalCoord(i);
			const float value = acc.getValue(voxelCoord);
			const size_t index = atomicAdd(voxelCount, 1);
			d_coords[index] = voxelCoord;
			d_values[index] = value;
		}
	});

	// Download the count of valid voxels
	size_t h_count;
	cudaCheck(hipMemcpy(&h_count, voxelCount, sizeof(size_t), hipMemcpyDeviceToHost));

	// Check if h_count exceeds allocated numVoxels
	if (h_count > numVoxels) {
		printf("Error: h_count exceeds allocated space\n");
		h_count = numVoxels;  // Adjust to prevent overflow
	}

	// Now you know how many valid voxels were processed and can download the data
	cudaCheck(hipMemcpy(h_coords, d_coords, h_count * sizeof(nanovdb::Coord), hipMemcpyDeviceToHost));
	cudaCheck(hipMemcpy(h_values, d_values, h_count * sizeof(float), hipMemcpyDeviceToHost));

	count = h_count;

	// Free allocated memory
	cudaCheck(hipFree(voxelCount));
	cudaCheck(hipFree(d_coords));
	cudaCheck(hipFree(d_values));
}

extern "C" void vel_thrust_kernel(nanovdb::Vec3fGrid* deviceGrid, const nanovdb::Vec3fGrid* velGrid,
                                  const uint64_t leafCount, const float voxelSize, const float dt, hipStream_t stream) {
	constexpr unsigned int numThreads = 256;
	const unsigned int numVoxels = 512 * leafCount;
	const unsigned int numBlocks = blocksPerGrid(numVoxels, numThreads);

	lambdaKernel<<<numBlocks, numThreads, 0, stream>>>(numVoxels, [deviceGrid, velGrid, voxelSize,
	                                                    dt] __device__(const uint64_t n) {
		auto& dtree = deviceGrid->tree();
		const auto& vtree = velGrid->tree();

		auto* leaf_d = dtree.getFirstNode<0>() + (n >> 9);
		const int i_d = n & 511;

		const auto* leaf_v = vtree.getFirstNode<0>() + (n >> 9);

		const auto velAccessor = velGrid->getAccessor();
		const auto velSampler = nanovdb::createSampler<1>(velAccessor);

		if (leaf_v->isActive()) {
			// Get the position of the voxel in index space
			const nanovdb::Coord voxelCoord = leaf_v->offsetToGlobalCoord(i_d);
			const nanovdb::Vec3f voxelCoordf = voxelCoord.asVec3s();
			const nanovdb::Vec3f velocity = velSampler(voxelCoordf);

			// Forward step
			const nanovdb::Vec3f forward_pos = voxelCoordf - velocity * (dt / voxelSize);
			const nanovdb::Vec3f v_forward = velSampler(forward_pos);

			// Backward step
			const nanovdb::Vec3f back_pos = voxelCoordf + velSampler(forward_pos) * (dt / voxelSize);
			const nanovdb::Vec3f v_backward = velSampler(back_pos);

			// Error estimation and correction
			const nanovdb::Vec3f error = 0.5f * (velocity - v_backward);
			nanovdb::Vec3f v_corrected = v_forward + error;

			// Limit the correction based on the neighborhood of the forward position
			const auto max_correction = nanovdb::Vec3f(cuda::std::abs(0.5f * (v_forward[0] - velocity[0])),
			                                           cuda::std::abs(0.5f * (v_forward[1] - velocity[1])),
			                                           cuda::std::abs(0.5f * (v_forward[2] - velocity[2])));
			v_corrected[0] =
			    cuda::std::clamp(v_corrected[0], v_forward[0] - max_correction[0], v_forward[0] + max_correction[0]);
			v_corrected[1] =
			    cuda::std::clamp(v_corrected[1], v_forward[1] - max_correction[1], v_forward[1] + max_correction[1]);
			v_corrected[2] =
			    cuda::std::clamp(v_corrected[2], v_forward[2] - max_correction[2], v_forward[2] + max_correction[2]);

			// Final advection (blend between semi-Lagrangian and BFECC result)
			constexpr float blend_factor = 0.8f;  // Adjust this value between 0 and 1
			nanovdb::Vec3f new_velocity;
			new_velocity[0] = lerp(v_forward[0], v_corrected[0], blend_factor);
			new_velocity[1] = lerp(v_forward[1], v_corrected[1], blend_factor);
			new_velocity[2] = lerp(v_forward[2], v_corrected[2], blend_factor);

			// Set the new velocity value
			leaf_d->setValue(voxelCoord, new_velocity);
		}
	});
}

extern "C" void thrust_kernel(nanovdb::FloatGrid* tempGrid, nanovdb::FloatGrid* deviceGrid, const nanovdb::Vec3fGrid* velGrid, const size_t leafCount,
                              const float voxelSize, const float dt, hipStream_t stream) {
	constexpr unsigned int numThreads = 256;
	const unsigned int numVoxels = 512 * leafCount;
	const unsigned int numBlocks = blocksPerGrid(numVoxels, numThreads);

	lambdaKernel<<<numBlocks, numThreads, 0, stream>>>(numVoxels, [tempGrid, deviceGrid, velGrid, voxelSize, dt] __device__(const size_t n) {
		auto& dtree = deviceGrid->tree();
		auto& vtree = velGrid->tree();
		auto& temp_tree = tempGrid->tree();

		const auto* leaf_temp = temp_tree.getFirstNode<0>() + (n >> 9);
		auto* leaf_d = dtree.getFirstNode<0>() + (n >> 9);
		auto* leaf_v = vtree.getFirstNode<0>() + (n >> 9);

		const int i_d = n & 511;
		const auto velAccessor = velGrid->getAccessor();
		const auto denAccessor = tempGrid->getAccessor();
		const auto velSampler = nanovdb::createSampler<1>(velAccessor);
		const auto denSampler = nanovdb::createSampler<1>(denAccessor);

		if (leaf_v->isActive()) {
			// Get the position of the voxel in index space
			const nanovdb::Coord voxelCoord = leaf_temp->offsetToGlobalCoord(i_d);
			const nanovdb::Vec3f voxelCoordf = voxelCoord.asVec3s();
			const float density = denSampler(voxelCoordf);

			// Forward step
			const nanovdb::Vec3f forward_pos = voxelCoordf - velSampler(voxelCoordf) * (dt / voxelSize);
			const float d_forward = denSampler(forward_pos);

			// Backward step
			const nanovdb::Vec3f back_pos = voxelCoordf + velSampler(forward_pos) * (dt / voxelSize);
			const float d_backward = denSampler(back_pos);

			// Error estimation and correction
			const float error = 0.5f * (density - d_backward);
			float d_corrected = d_forward + error;

			// Limit the correction based on the neighborhood of the forward position
			const float max_correction = 0.5f * cuda::std::fabs(d_forward - density);
			d_corrected = cuda::std::clamp(d_corrected, d_forward - max_correction, d_forward + max_correction);

			// Final advection (blend between semi-Lagrangian and BFECC result)
			constexpr float blend_factor = 0.8f;  // Adjust this value between 0 and 1
			float new_density = lerp(d_forward, d_corrected, blend_factor);

			// Ensure non-negativity
			new_density = cuda::std::fmax(0.0f, new_density);

			// Set the new density value
			leaf_d->setValue(voxelCoord, new_density);
		}
	});
	cudaCheckError();
}

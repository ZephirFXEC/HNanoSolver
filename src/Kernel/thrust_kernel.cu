#include "hip/hip_runtime.h"
#include <cuda/std/__algorithm/clamp.h>
#include <nanovdb/NanoVDB.h>
#include <nanovdb/util/SampleFromVoxels.h>

#include <cuda/std/cmath>
#include <nanovdb/util/cuda/CudaPointsToGrid.cuh>

#include "../Utils/GridData.hpp"
#include "utils.cuh"


extern "C" void advect_points_to_grid_f(const OpenFloatGrid& in_data, const nanovdb::Vec3fGrid* vel_grid,
                                        NanoFloatGrid& out_data, const float voxelSize, const float dt,
                                        const hipStream_t& stream) {
	const size_t npoints = in_data.size;

	// Allocate and copy coordinates to the device
	nanovdb::Coord* d_coords = nullptr;
	float* d_values = nullptr;
	hipMalloc(&d_coords, npoints * sizeof(nanovdb::Coord));
	hipMalloc(&d_values, npoints * sizeof(float));

	hipMemcpyAsync(d_coords, (nanovdb::Coord*)in_data.pCoords, npoints * sizeof(nanovdb::Coord),
	                hipMemcpyHostToDevice, stream);

	hipMemcpyAsync(d_values, in_data.pValues, npoints * sizeof(float), hipMemcpyHostToDevice, stream);

	float* temp_values = nullptr;
	hipMalloc(&temp_values, npoints * sizeof(float));

	// Generate a NanoVDB grid that contains the list of voxels on the device
	nanovdb::GridHandle<nanovdb::CudaDeviceBuffer> handle =
	    nanovdb::cudaVoxelsToGrid<float>(d_coords, npoints, voxelSize);
	nanovdb::FloatGrid* d_grid = handle.deviceGrid<float>();

	constexpr unsigned int numThreads = 256;
	const unsigned int numBlocks = blocksPerGrid(npoints, numThreads);

	lambdaKernel<<<numBlocks, numThreads, 0, stream>>>(npoints, [=] __device__(const size_t tid) {
		d_grid->tree().set<nanovdb::SetVoxel<float>>(d_coords[tid], d_values[tid]);
	});


	lambdaKernel<<<numBlocks, numThreads, 0, stream>>>(npoints, [=] __device__(const size_t tid) {
		const nanovdb::Coord& ijk = d_coords[tid];
		const float& density = d_values[tid];

		const auto velAccessor = vel_grid->getAccessor();
		const auto denAccessor = d_grid->getAccessor();
		const auto velSampler = nanovdb::createSampler<1>(velAccessor);
		const auto denSampler = nanovdb::createSampler<1>(denAccessor);

		const nanovdb::Vec3f voxelCoordf = ijk.asVec3s();

		// Forward step
		const nanovdb::Vec3f forward_pos = voxelCoordf - velSampler(voxelCoordf) * (dt / voxelSize);
		const float d_forward = denSampler(forward_pos);

		// Backward step
		const nanovdb::Vec3f back_pos = voxelCoordf + velSampler(forward_pos) * (dt / voxelSize);
		const float d_backward = denSampler(back_pos);

		// Error estimation and correction
		const float error = 0.5f * (density - d_backward);
		float d_corrected = d_forward + error;

		// Limit the correction based on the neighborhood of the forward position
		const float max_correction = 0.5f * cuda::std::fabs(d_forward - density);
		d_corrected = cuda::std::clamp(d_corrected, d_forward - max_correction, d_forward + max_correction);

		// Final advection (blend between semi-Lagrangian and BFECC result)
		constexpr float blend_factor = 0.8f;  // Adjust this value between 0 and 1
		float new_density = lerp(d_forward, d_corrected, blend_factor);

		// Ensure non-negativity
		new_density = cuda::std::fmax(0.0f, new_density);

		temp_values[tid] = new_density;
	});

	out_data.size = npoints;
	out_data.pCoords = new nanovdb::Coord[out_data.size];
	out_data.pValues = new float[out_data.size];

	// Copy results back to the host
	hipMemcpyAsync(out_data.pValues, temp_values, sizeof(float) * npoints, hipMemcpyDeviceToHost, stream);
	hipMemcpyAsync(out_data.pCoords, d_coords, sizeof(nanovdb::Coord) * npoints, hipMemcpyDeviceToHost, stream);

	hipFree(d_coords);
	hipFree(d_values);
	hipFree(temp_values);
}


extern "C" void advect_points_to_grid_v(const OpenVectorGrid& in_data, NanoVectorGrid& out_data, const float voxelSize,
                                        const float dt, const hipStream_t& stream) {
	const size_t npoints = in_data.size;

	// Allocate and copy coordinates to the device
	nanovdb::Coord* d_coords = nullptr;
	nanovdb::Vec3f* d_values = nullptr;
	hipMalloc(&d_coords, npoints * sizeof(nanovdb::Coord));
	hipMalloc(&d_values, npoints * sizeof(nanovdb::Vec3f));

	hipMemcpyAsync(d_coords, (nanovdb::Coord*)in_data.pCoords, npoints * sizeof(nanovdb::Coord),
	                hipMemcpyHostToDevice, stream);

	hipMemcpyAsync(d_values, (nanovdb::Vec3f*)in_data.pValues, npoints * sizeof(nanovdb::Vec3f),
	                hipMemcpyHostToDevice, stream);

	nanovdb::Vec3f* temp_values = nullptr;
	hipMalloc(&temp_values, npoints * sizeof(nanovdb::Vec3f));

	// Generate a NanoVDB grid that contains the list of voxels on the device
	nanovdb::GridHandle<nanovdb::CudaDeviceBuffer> handle =
	    nanovdb::cudaVoxelsToGrid<nanovdb::Vec3f>(d_coords, npoints, voxelSize);
	nanovdb::Vec3fGrid* d_grid = handle.deviceGrid<nanovdb::Vec3f>();


	constexpr unsigned int numThreads = 256;
	const unsigned int numBlocks = blocksPerGrid(npoints, numThreads);

	lambdaKernel<<<numBlocks, numThreads, 0, stream>>>(npoints, [=] __device__(const size_t tid) {
		const nanovdb::Coord& ijk = d_coords[tid];
		const nanovdb::Vec3f& velocity = d_values[tid];

		d_grid->tree().set<nanovdb::SetVoxel<nanovdb::Vec3f>>(ijk, velocity);

		const auto velAccessor = d_grid->getAccessor();
		const auto velSampler = nanovdb::createSampler<1>(velAccessor);

		// Perform forward and backward advection using velocity
		const nanovdb::Vec3f voxelCoordf = ijk.asVec3s();
		const nanovdb::Vec3f forward_pos = voxelCoordf - velocity * (dt / voxelSize);
		const nanovdb::Vec3f backward_pos = voxelCoordf + velocity * (dt / voxelSize);

		const nanovdb::Vec3f v_forward = velSampler(forward_pos);
		const nanovdb::Vec3f v_backward = velSampler(backward_pos);

		// Error estimation and correction
		const nanovdb::Vec3f error = 0.5f * (velocity - v_backward);
		nanovdb::Vec3f v_corrected = v_forward + error;

		nanovdb::Vec3f max_correction;
		max_correction[0] = cuda::std::abs(0.5f * (v_forward[0] - velocity[0]));
		max_correction[1] = cuda::std::abs(0.5f * (v_forward[1] - velocity[1]));
		max_correction[2] = cuda::std::abs(0.5f * (v_forward[2] - velocity[2]));

		v_corrected[0] =
		    cuda::std::clamp(v_corrected[0], v_forward[0] - max_correction[0], v_forward[0] + max_correction[0]);
		v_corrected[1] =
		    cuda::std::clamp(v_corrected[1], v_forward[1] - max_correction[1], v_forward[1] + max_correction[1]);
		v_corrected[2] =
		    cuda::std::clamp(v_corrected[2], v_forward[2] - max_correction[2], v_forward[2] + max_correction[2]);


		constexpr float blend_factor = 0.8f;  // Adjust this value between 0 and 1
		nanovdb::Vec3f new_velocity;
		new_velocity[0] = lerp(v_forward[0], v_corrected[0], blend_factor);
		new_velocity[1] = lerp(v_forward[1], v_corrected[1], blend_factor);
		new_velocity[2] = lerp(v_forward[2], v_corrected[2], blend_factor);

		// Store the new velocity and voxel position
		temp_values[tid] = new_velocity;
	});

	out_data.size = npoints;
	out_data.pCoords = new nanovdb::Coord[out_data.size];
	out_data.pValues = new nanovdb::Vec3f[out_data.size];

	// Copy results back to the host
	hipMemcpyAsync(out_data.pValues, temp_values, sizeof(nanovdb::Vec3f) * npoints, hipMemcpyDeviceToHost, stream);
	hipMemcpyAsync(out_data.pCoords, d_coords, sizeof(nanovdb::Coord) * npoints, hipMemcpyDeviceToHost, stream);

	hipFree(d_coords);
	hipFree(d_values);
	hipFree(temp_values);
}


extern "C" void vel_thrust_kernel(const nanovdb::Vec3fGrid* velGrid, const uint64_t leafCount, const float voxelSize,
                                  const float dt, hipStream_t stream, nanovdb::Coord* h_coords,
                                  nanovdb::Vec3f* h_values, size_t& count) {
	size_t* voxelCount = nullptr;
	cudaCheck(hipMalloc(&voxelCount, sizeof(size_t)));
	cudaCheck(hipMemset(voxelCount, 0, sizeof(size_t)));

	constexpr unsigned int numThreads = 256;
	const unsigned int numVoxels = 512 * leafCount;
	const unsigned int numBlocks = blocksPerGrid(numVoxels, numThreads);

	nanovdb::Coord* d_coords = nullptr;
	nanovdb::Vec3f* d_values = nullptr;

	cudaCheck(hipMalloc(&d_coords, numVoxels * sizeof(nanovdb::Coord)));
	cudaCheck(hipMalloc(&d_values, numVoxels * sizeof(nanovdb::Vec3f)));
	cudaCheck(hipMemset(d_coords, 0, numVoxels * sizeof(nanovdb::Coord)));
	cudaCheck(hipMemset(d_values, 0, numVoxels * sizeof(nanovdb::Vec3f)));

	hipDeviceSynchronize();

	lambdaKernel<<<numBlocks, numThreads, 0, stream>>>(numVoxels, [velGrid, voxelSize, dt, voxelCount, d_coords,
	                                                               d_values] __device__(const uint64_t n) {
		const auto& vtree = velGrid->tree();
		const uint32_t i_d = n & 511;
		const auto* leaf_v = vtree.getFirstNode<0>() + (n >> 9);
		const auto velAccessor = velGrid->getAccessor();
		const auto velSampler = nanovdb::createSampler<1>(velAccessor);

		if (leaf_v->isActive()) {
			// Get the position of the voxel in index space
			const nanovdb::Coord voxelCoord = leaf_v->offsetToGlobalCoord(i_d);
			const nanovdb::Vec3f voxelCoordf = voxelCoord.asVec3s();
			const nanovdb::Vec3f velocity = velSampler(voxelCoordf);

			// Forward step
			const nanovdb::Vec3f forward_pos = voxelCoordf - velocity * (dt / voxelSize);
			const nanovdb::Vec3f v_forward = velSampler(forward_pos);

			// Backward step
			const nanovdb::Vec3f back_pos = voxelCoordf + velSampler(forward_pos) * (dt / voxelSize);
			const nanovdb::Vec3f v_backward = velSampler(back_pos);

			// Error estimation and correction
			const nanovdb::Vec3f error = 0.5f * (velocity - v_backward);
			nanovdb::Vec3f v_corrected = v_forward + error;

			// Limit the correction based on the neighborhood of the forward position
			const auto max_correction = nanovdb::Vec3f(cuda::std::abs(0.5f * (v_forward[0] - velocity[0])),
			                                           cuda::std::abs(0.5f * (v_forward[1] - velocity[1])),
			                                           cuda::std::abs(0.5f * (v_forward[2] - velocity[2])));
			v_corrected[0] =
			    cuda::std::clamp(v_corrected[0], v_forward[0] - max_correction[0], v_forward[0] + max_correction[0]);
			v_corrected[1] =
			    cuda::std::clamp(v_corrected[1], v_forward[1] - max_correction[1], v_forward[1] + max_correction[1]);
			v_corrected[2] =
			    cuda::std::clamp(v_corrected[2], v_forward[2] - max_correction[2], v_forward[2] + max_correction[2]);

			// Final advection (blend between semi-Lagrangian and BFECC result)
			constexpr float blend_factor = 0.8f;  // Adjust this value between 0 and 1
			nanovdb::Vec3f new_velocity;
			new_velocity[0] = lerp(v_forward[0], v_corrected[0], blend_factor);
			new_velocity[1] = lerp(v_forward[1], v_corrected[1], blend_factor);
			new_velocity[2] = lerp(v_forward[2], v_corrected[2], blend_factor);

			// TODO: Remove that and create another kenrel to handle the data copy to avoid the usage of atomicAdd
			const size_t index = atomicAdd(voxelCount, 1);
			d_coords[index] = voxelCoord;
			d_values[index] = new_velocity;
		}
	});

	hipDeviceSynchronize();

	// Download the count of valid voxels
	size_t h_count;
	cudaCheck(hipMemcpy(&h_count, voxelCount, sizeof(size_t), hipMemcpyDeviceToHost));

	// Now you know how many valid voxels were processed and can download the data
	cudaCheck(hipMemcpy(h_coords, d_coords, h_count * sizeof(nanovdb::Coord), hipMemcpyDeviceToHost));
	cudaCheck(hipMemcpy(h_values, d_values, h_count * sizeof(nanovdb::Vec3f), hipMemcpyDeviceToHost));

	count = h_count;

	// Free allocated memory
	cudaCheck(hipFree(voxelCount));
	cudaCheck(hipFree(d_coords));
	cudaCheck(hipFree(d_values));
}

extern "C" void thrust_kernel(const nanovdb::FloatGrid* deviceGrid, const nanovdb::Vec3fGrid* velGrid,
                              const size_t leafCount, const float voxelSize, const float dt, hipStream_t stream,
                              nanovdb::Coord* h_coords, float* h_values, size_t& count) {
	size_t* voxelCount = nullptr;
	cudaCheck(hipMalloc(&voxelCount, sizeof(size_t)));
	cudaCheck(hipMemset(voxelCount, 0, sizeof(size_t)));

	constexpr unsigned int numThreads = 256;
	const unsigned int numVoxels = 512 * leafCount;
	const unsigned int numBlocks = blocksPerGrid(numVoxels, numThreads);

	nanovdb::Coord* d_coords = nullptr;
	float* d_values = nullptr;

	cudaCheck(hipMalloc(&d_coords, numVoxels * sizeof(nanovdb::Coord)));
	cudaCheck(hipMalloc(&d_values, numVoxels * sizeof(float)));
	cudaCheck(hipMemset(d_coords, 0, numVoxels * sizeof(nanovdb::Coord)));
	cudaCheck(hipMemset(d_values, 0, numVoxels * sizeof(float)));

	hipDeviceSynchronize();

	lambdaKernel<<<numBlocks, numThreads, 0, stream>>>(
	    numVoxels, [deviceGrid, velGrid, voxelSize, dt, voxelCount, d_coords, d_values] __device__(const size_t n) {
		    const auto& dtree = deviceGrid->tree();

		    const auto* leaf_d = dtree.getFirstNode<0>() + (n >> 9);

		    const int i_d = n & 511;
		    const auto velAccessor = velGrid->getAccessor();
		    const auto denAccessor = deviceGrid->getAccessor();
		    const auto velSampler = nanovdb::createSampler<1>(velAccessor);
		    const auto denSampler = nanovdb::createSampler<1>(denAccessor);

		    if (leaf_d->isActive()) {
			    // Get the position of the voxel in index space
			    const nanovdb::Coord voxelCoord = leaf_d->offsetToGlobalCoord(i_d);
			    const nanovdb::Vec3f voxelCoordf = voxelCoord.asVec3s();
			    const float density = denSampler(voxelCoordf);

			    // Forward step
			    const nanovdb::Vec3f forward_pos = voxelCoordf - velSampler(voxelCoordf) * (dt / voxelSize);
			    const float d_forward = denSampler(forward_pos);

			    // Backward step
			    const nanovdb::Vec3f back_pos = voxelCoordf + velSampler(forward_pos) * (dt / voxelSize);
			    const float d_backward = denSampler(back_pos);

			    // Error estimation and correction
			    const float error = 0.5f * (density - d_backward);
			    float d_corrected = d_forward + error;

			    // Limit the correction based on the neighborhood of the forward position
			    const float max_correction = 0.5f * cuda::std::fabs(d_forward - density);
			    d_corrected = cuda::std::clamp(d_corrected, d_forward - max_correction, d_forward + max_correction);

			    // Final advection (blend between semi-Lagrangian and BFECC result)
			    constexpr float blend_factor = 0.8f;  // Adjust this value between 0 and 1
			    float new_density = lerp(d_forward, d_corrected, blend_factor);

			    // Ensure non-negativity
			    new_density = cuda::std::fmax(0.0f, new_density);

			    // TODO: Remove that and create another kenrel to handle the data copy to avoid the usage of atomicAdd
			    const size_t index = atomicAdd(voxelCount, 1);
			    d_coords[index] = voxelCoord;
			    d_values[index] = new_density;
		    }
	    });
	cudaCheckError();

	hipDeviceSynchronize();

	// Download the count of valid voxels
	size_t h_count;
	cudaCheck(hipMemcpy(&h_count, voxelCount, sizeof(size_t), hipMemcpyDeviceToHost));

	// Now you know how many valid voxels were processed and can download the data
	cudaCheck(hipMemcpy(h_coords, d_coords, h_count * sizeof(nanovdb::Coord), hipMemcpyDeviceToHost));
	cudaCheck(hipMemcpy(h_values, d_values, h_count * sizeof(float), hipMemcpyDeviceToHost));

	count = h_count;

	// Free allocated memory
	cudaCheck(hipFree(voxelCount));
	cudaCheck(hipFree(d_coords));
	cudaCheck(hipFree(d_values));
}

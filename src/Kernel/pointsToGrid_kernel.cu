#include <nanovdb/util/GridBuilder.h>
#include <nanovdb/util/cuda/CudaPointsToGrid.cuh>

#include "utils.cuh"
#include "../Utils/GridData.hpp"


extern "C" void pointToGrid(const std::vector<nanovdb::Coord>& coords, const std::vector<float>& values, const float voxelSize, NanoFloatGrid& out_data) {


	const size_t npoints = coords.size();
	nanovdb::Coord* d_coords = nullptr;
	cudaCheck(hipMalloc(&d_coords, npoints * sizeof(nanovdb::Coord)));
	cudaCheck(hipMemcpyAsync(d_coords, coords.data(), npoints * sizeof(nanovdb::Coord), hipMemcpyHostToDevice));

	// Generate a NanoVDB grid that contains the list of voxels on the device
	nanovdb::GridHandle<nanovdb::CudaDeviceBuffer> handle = nanovdb::cudaVoxelsToGrid<float>(d_coords, npoints, voxelSize);
	nanovdb::FloatGrid* d_grid = handle.deviceGrid<float>();

	// Define a list of values and copy them to the device
	float *d_values;
	cudaCheck(hipMalloc(&d_values, npoints * sizeof(float)));
	cudaCheck(hipMemcpyAsync(d_values, values.data(), npoints * sizeof(float), hipMemcpyHostToDevice));

	// Synchronize to ensure all data is copied before launching the kernel
	cudaCheck(hipDeviceSynchronize());

	// Launch a device kernel that sets the values of voxels define above and prints them
	constexpr unsigned int numThreads = 256;
	const unsigned int numBlocks = blocksPerGrid(npoints, numThreads);
	using OpT = nanovdb::SetVoxel<float>;

	lambdaKernel<<<numBlocks, numThreads>>>(npoints, [=] __device__(const size_t tid) {
		const nanovdb::Coord &ijk = d_coords[tid];
		d_grid->tree().set<OpT>(ijk, d_values[tid]);// normally one should use a ValueAccessor
	}); cudaCheckError();


	cudaCheck(hipMalloc(&out_data.pValues, sizeof(float) * npoints));
	cudaCheck(hipMalloc(&out_data.pCoords, sizeof(nanovdb::Coord) * npoints));
	cudaCheck(hipMemcpyAsync(out_data.pValues, d_values, sizeof(float), hipMemcpyDeviceToHost));
	cudaCheck(hipMemcpyAsync(out_data.pCoords, d_grid, sizeof(nanovdb::Coord), hipMemcpyDeviceToHost));

	// free arrays allocated on the device
	cudaCheck(hipFree(d_coords));
	cudaCheck(hipFree(d_values));
}
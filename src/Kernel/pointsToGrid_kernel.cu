#include <nanovdb/util/GridBuilder.h>
#include <nanovdb/util/cuda/CudaPointsToGrid.cuh>

#include "utils.cuh"

struct Grid {
	std::vector<nanovdb::Coord> coords{};
	std::vector<float> values{};
	float voxelSize = 0.5f;
};

extern "C" void pointToGrid(const Grid& gridData, nanovdb::GridHandle<nanovdb::CudaDeviceBuffer>& out_handle) {

	const size_t npoints = gridData.coords.size();
	nanovdb::Coord* d_coords = nullptr;
	cudaCheck(hipMalloc(&d_coords, npoints * sizeof(nanovdb::Coord)));
	cudaCheck(hipMemcpyAsync(d_coords, gridData.coords.data(), npoints * sizeof(nanovdb::Coord), hipMemcpyHostToDevice));

	// Generate a NanoVDB grid that contains the list of voxels on the device
	out_handle = nanovdb::cudaVoxelsToGrid<float>(d_coords, npoints, gridData.voxelSize);
	nanovdb::FloatGrid* d_grid = out_handle.deviceGrid<float>();

	// Define a list of values and copy them to the device
	float *d_values;
	cudaCheck(hipMalloc(&d_values, npoints * sizeof(float)));
	cudaCheck(hipMemcpyAsync(d_values, gridData.values.data(), npoints * sizeof(float), hipMemcpyHostToDevice));

	// Synchronize to ensure all data is copied before launching the kernel
	cudaCheck(hipDeviceSynchronize());

	// Launch a device kernel that sets the values of voxels define above and prints them
	constexpr unsigned int numThreads = 256;
	const unsigned int numBlocks = blocksPerGrid(npoints, numThreads);
	using OpT = nanovdb::SetVoxel<float>;

	lambdaKernel<<<numBlocks, numThreads>>>(npoints, [=] __device__(const size_t tid) {
		const nanovdb::Coord &ijk = d_coords[tid];
		d_grid->tree().set<OpT>(ijk, d_values[tid]);// normally one should use a ValueAccessor
	}); cudaCheckError();

	// free arrays allocated on the device
	cudaCheck(hipFree(d_coords));
	cudaCheck(hipFree(d_values));
}